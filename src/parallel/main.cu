#include "hip/hip_runtime.h"
/**
 * @file main.c
 * @author Hollweck, Wigg
 * @date 10 April 2019
 * @brief Main file.
 *
 * Here typically goes a more extensive explanation of what the header
 * defines. Doxygens tags are words preceeded by either a backslash @\
 * or by an at symbol @@.
 * @see http://www.stack.nl/~dimitri/doxygen/docblocks.html
 * @see http://www.stack.nl/~dimitri/doxygen/commands.html
 */

#include "memory.h"
#include "io.h"
#include "integration.h"
#include "boundaries.h"
#include <time.h>
#include <math.h>
#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <vector>
#include <numeric>
#include <cmath>
int BLOCK_SIZE = -1; // Default value, can be overridden by command line

typedef struct{
    int i;
    int j;
    int position;
} BoundaryPoint;


// Macro para verificar erros CUDA
#define CHECK_CUDA_ERROR(call) \
    do { \
        hipError_t err = call; \
        if (err != hipSuccess) { \
            fprintf(stderr, "CUDA error at %s:%d - %s\n", __FILE__, __LINE__, hipGetErrorString(err)); \
            exit(EXIT_FAILURE); \
        } \
    } while(0)

// Replace the existing allocate_unified_memory function with this device memory version
int allocate_device_memory(double ***u, double ***v, double ***p, double ***res, double ***RHS, double ***F, double ***G, 
                          int i_max, int j_max, BoundaryPoint **borders, int num_border_points) {
    int rows = i_max + 2;
    int cols = j_max + 2;
    
    // Allocate host-side array pointers
    *u = (double**)malloc(rows * sizeof(double*));
    *v = (double**)malloc(rows * sizeof(double*));
    *p = (double**)malloc(rows * sizeof(double*));
    *res = (double**)malloc(rows * sizeof(double*));
    *RHS = (double**)malloc(rows * sizeof(double*));
    *F = (double**)malloc(rows * sizeof(double*));
    *G = (double**)malloc(rows * sizeof(double*));
    
    // Allocate device-side array pointers
    double **d_u, **d_v, **d_p, **d_res, **d_RHS, **d_F, **d_G;
    CHECK_CUDA_ERROR(hipMalloc((void**)&d_u, rows * sizeof(double*)));
    CHECK_CUDA_ERROR(hipMalloc((void**)&d_v, rows * sizeof(double*)));
    CHECK_CUDA_ERROR(hipMalloc((void**)&d_p, rows * sizeof(double*)));
    CHECK_CUDA_ERROR(hipMalloc((void**)&d_res, rows * sizeof(double*)));
    CHECK_CUDA_ERROR(hipMalloc((void**)&d_RHS, rows * sizeof(double*)));
    CHECK_CUDA_ERROR(hipMalloc((void**)&d_F, rows * sizeof(double*)));
    CHECK_CUDA_ERROR(hipMalloc((void**)&d_G, rows * sizeof(double*)));
    
    // Allocate device memory for borders
    CHECK_CUDA_ERROR(hipMalloc((void**)borders, num_border_points * sizeof(BoundaryPoint)));
    
    // Allocate device memory for data
    double *d_u_data, *d_v_data, *d_p_data, *d_res_data, *d_RHS_data, *d_F_data, *d_G_data;
    CHECK_CUDA_ERROR(hipMalloc((void**)&d_u_data, rows * cols * sizeof(double)));
    CHECK_CUDA_ERROR(hipMalloc((void**)&d_v_data, rows * cols * sizeof(double)));
    CHECK_CUDA_ERROR(hipMalloc((void**)&d_p_data, rows * cols * sizeof(double)));
    CHECK_CUDA_ERROR(hipMalloc((void**)&d_res_data, rows * cols * sizeof(double)));
    CHECK_CUDA_ERROR(hipMalloc((void**)&d_RHS_data, rows * cols * sizeof(double)));
    CHECK_CUDA_ERROR(hipMalloc((void**)&d_F_data, rows * cols * sizeof(double)));
    CHECK_CUDA_ERROR(hipMalloc((void**)&d_G_data, rows * cols * sizeof(double)));
    
    // Temporary host arrays for initialization
    double *h_u_data = (double*)calloc(rows * cols, sizeof(double));
    double *h_v_data = (double*)calloc(rows * cols, sizeof(double));
    double *h_p_data = (double*)calloc(rows * cols, sizeof(double));
    double *h_res_data = (double*)calloc(rows * cols, sizeof(double));
    double *h_RHS_data = (double*)calloc(rows * cols, sizeof(double));
    double *h_F_data = (double*)calloc(rows * cols, sizeof(double));
    double *h_G_data = (double*)calloc(rows * cols, sizeof(double));
    
    // Host pointers to device data (for kernels)
    double **h_device_ptrs[7];
    h_device_ptrs[0] = (double**)malloc(rows * sizeof(double*));
    h_device_ptrs[1] = (double**)malloc(rows * sizeof(double*));
    h_device_ptrs[2] = (double**)malloc(rows * sizeof(double*));
    h_device_ptrs[3] = (double**)malloc(rows * sizeof(double*));
    h_device_ptrs[4] = (double**)malloc(rows * sizeof(double*));
    h_device_ptrs[5] = (double**)malloc(rows * sizeof(double*));
    h_device_ptrs[6] = (double**)malloc(rows * sizeof(double*));
    
    // Setup row pointers
    for (int i = 0; i < rows; i++) {
        h_device_ptrs[0][i] = d_u_data + i * cols;
        h_device_ptrs[1][i] = d_v_data + i * cols;
        h_device_ptrs[2][i] = d_p_data + i * cols;
        h_device_ptrs[3][i] = d_res_data + i * cols;
        h_device_ptrs[4][i] = d_RHS_data + i * cols;
        h_device_ptrs[5][i] = d_F_data + i * cols;
        h_device_ptrs[6][i] = d_G_data + i * cols;
    }
    
    // Copy the pointers to device
    CHECK_CUDA_ERROR(hipMemcpy(d_u, h_device_ptrs[0], rows * sizeof(double*), hipMemcpyHostToDevice));
    CHECK_CUDA_ERROR(hipMemcpy(d_v, h_device_ptrs[1], rows * sizeof(double*), hipMemcpyHostToDevice));
    CHECK_CUDA_ERROR(hipMemcpy(d_p, h_device_ptrs[2], rows * sizeof(double*), hipMemcpyHostToDevice));
    CHECK_CUDA_ERROR(hipMemcpy(d_res, h_device_ptrs[3], rows * sizeof(double*), hipMemcpyHostToDevice));
    CHECK_CUDA_ERROR(hipMemcpy(d_RHS, h_device_ptrs[4], rows * sizeof(double*), hipMemcpyHostToDevice));
    CHECK_CUDA_ERROR(hipMemcpy(d_F, h_device_ptrs[5], rows * sizeof(double*), hipMemcpyHostToDevice));
    CHECK_CUDA_ERROR(hipMemcpy(d_G, h_device_ptrs[6], rows * sizeof(double*), hipMemcpyHostToDevice));
    
    // Copy zeros to device arrays
    CHECK_CUDA_ERROR(hipMemcpy(d_u_data, h_u_data, rows * cols * sizeof(double), hipMemcpyHostToDevice));
    CHECK_CUDA_ERROR(hipMemcpy(d_v_data, h_v_data, rows * cols * sizeof(double), hipMemcpyHostToDevice));
    CHECK_CUDA_ERROR(hipMemcpy(d_p_data, h_p_data, rows * cols * sizeof(double), hipMemcpyHostToDevice));
    CHECK_CUDA_ERROR(hipMemcpy(d_res_data, h_res_data, rows * cols * sizeof(double), hipMemcpyHostToDevice));
    CHECK_CUDA_ERROR(hipMemcpy(d_RHS_data, h_RHS_data, rows * cols * sizeof(double), hipMemcpyHostToDevice));
    CHECK_CUDA_ERROR(hipMemcpy(d_F_data, h_F_data, rows * cols * sizeof(double), hipMemcpyHostToDevice));
    CHECK_CUDA_ERROR(hipMemcpy(d_G_data, h_G_data, rows * cols * sizeof(double), hipMemcpyHostToDevice));
    
    // Store device pointers for kernel calls
    (*u) = d_u;
    (*v) = d_v;
    (*p) = d_p;
    (*res) = d_res;
    (*RHS) = d_RHS;
    (*F) = d_F;
    (*G) = d_G;
    
    // Free temporary host memory
    free(h_u_data);
    free(h_v_data);
    free(h_p_data);
    free(h_res_data);
    free(h_RHS_data);
    free(h_F_data);
    free(h_G_data);
    
    for (int i = 0; i < 7; i++) {
        free(h_device_ptrs[i]);
    }
    
    return 0;
}

// Update the free function to match
void free_device_memory(double **u, double **v, double **p, double **res, double **RHS, double **F, double **G, 
                       BoundaryPoint *borders) {
    // First row of each array contains the pointer to the contiguous data
    double *u_data, *v_data, *p_data, *res_data, *RHS_data, *F_data, *G_data;
    
    // Get the first data pointer from each array
    CHECK_CUDA_ERROR(hipMemcpy(&u_data, u, sizeof(double*), hipMemcpyDeviceToHost));
    CHECK_CUDA_ERROR(hipMemcpy(&v_data, v, sizeof(double*), hipMemcpyDeviceToHost));
    CHECK_CUDA_ERROR(hipMemcpy(&p_data, p, sizeof(double*), hipMemcpyDeviceToHost));
    CHECK_CUDA_ERROR(hipMemcpy(&res_data, res, sizeof(double*), hipMemcpyDeviceToHost));
    CHECK_CUDA_ERROR(hipMemcpy(&RHS_data, RHS, sizeof(double*), hipMemcpyDeviceToHost));
    CHECK_CUDA_ERROR(hipMemcpy(&F_data, F, sizeof(double*), hipMemcpyDeviceToHost));
    CHECK_CUDA_ERROR(hipMemcpy(&G_data, G, sizeof(double*), hipMemcpyDeviceToHost));
    
    // Free data memory
    hipFree(u_data);
    hipFree(v_data);
    hipFree(p_data);
    hipFree(res_data);
    hipFree(RHS_data);
    hipFree(F_data);
    hipFree(G_data);
    
    // Free pointer arrays
    hipFree(u);
    hipFree(v);
    hipFree(p);
    hipFree(res);
    hipFree(RHS);
    hipFree(F);
    hipFree(G);
    
    // Free border points
    hipFree(borders);
}


void precalculate_borders(int i_max, int j_max, BoundaryPoint *borders_ptr) {
    // Create temporary host array for border points
    BoundaryPoint *h_borders = (BoundaryPoint*)malloc(2 * (i_max + j_max + 2) * sizeof(BoundaryPoint));
    
    int index = 0;
    for (int i = 0; i <= i_max + 1; i++) {
        for (int j = 0; j <= j_max + 1; j++) {
            if (i == 0 || i == i_max + 1 || j == 0 || j == j_max + 1) {
                h_borders[index].i = i;
                h_borders[index].j = j;
                h_borders[index].position = (i == 0) ? LEFT : (i == i_max + 1) ? RIGHT : (j == 0) ? BOTTOM : TOP;
                index++;
            }
        }
    }
    
    // Copy border points to device
    CHECK_CUDA_ERROR(hipMemcpy(borders_ptr, h_borders, index * sizeof(BoundaryPoint), hipMemcpyHostToDevice));
    
    // Free temporary host memory
    free(h_borders);
}


// Kernels CUDA que podem acessar diretamente as matrizes 2D
__global__ void calculate_RHS_kernel(double **RHS, double **F, double **G, 
                                   int i_max, int j_max, double delta_t, 
                                   double delta_x, double delta_y) {
    int i = blockIdx.x * blockDim.x + threadIdx.x + 1;
    int j = blockIdx.y * blockDim.y + threadIdx.y + 1;
    
    if (i <= i_max && j <= j_max) {
        RHS[i][j] = 1.0 / delta_t * ((F[i][j] - F[i-1][j])/delta_x + 
                                     (G[i][j] - G[i][j-1])/delta_y);
    }
}

__global__ void update_velocities_kernel(double **u, double **v, double **F, double **G, double **p,
                                        int i_max, int j_max, double delta_t, 
                                        double delta_x, double delta_y) {
    int i = blockIdx.x * blockDim.x + threadIdx.x + 1;
    int j = blockIdx.y * blockDim.y + threadIdx.y + 1;
    
    if (i <= i_max && j <= j_max) {
        if (i <= i_max - 1) {
            u[i][j] = F[i][j] - delta_t * (p[i+1][j] - p[i][j]) / delta_x;
        }
        if (j <= j_max - 1) {
            v[i][j] = G[i][j] - delta_t * (p[i][j+1] - p[i][j]) / delta_y;
        }
    }
}


// Kernel otimizado para atualizar bordas usando pontos pré-calculados
__global__ void update_boundaries_with_precalc_kernel(double **p, BoundaryPoint *borders, int border_count) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    
    if (idx < border_count) {
        int i = borders[idx].i;
        int j = borders[idx].j;
        int position = borders[idx].position;
        
        // Aplicar condição de Neumann apropriada baseada na posição
        switch (position) {
            case LEFT:
                p[i][j] = p[i+1][j];  // Copia do vizinho à direita
                break;
            case RIGHT:
                p[i][j] = p[i-1][j];  // Copia do vizinho à esquerda
                break;
            case BOTTOM:
                p[i][j] = p[i][j+1];  // Copia do vizinho acima
                break;
            case TOP:
                p[i][j] = p[i][j-1];  // Copia do vizinho abaixo
                break;
        }
    }
}

// Função auxiliar no host para calcular a norma L2 de uma matriz UVA
double calculate_L2_norm_host_uva(double **matrix, int i_max, int j_max) {
    double norm_sq_sum = 0.0;
    if (i_max == 0 || j_max == 0) return 0.0;

    for (int r = 1; r <= i_max; r++) {
        for (int c = 1; c <= j_max; c++) {
            norm_sq_sum += matrix[r][c] * matrix[r][c];
        }
    }
    return sqrt(norm_sq_sum / (i_max * j_max));
}

// Device versions of differential functions
__device__ double du2_dx_device(double** u, double** v, int i, int j, double delta_x, double gamma) {
    double stencil1 = 0.5 * (u[i][j] + u[i+1][j]);
    double stencil2 = 0.5 * (u[i-1][j] + u[i][j]);

    double stencil3 = fabs(stencil1) * 0.5 * (u[i][j] - u[i+1][j]);
    double stencil4 = fabs(stencil2) * 0.5 * (u[i-1][j] - u[i][j]);

    return (1.0/delta_x) * (stencil1*stencil1 - stencil2*stencil2) + (gamma / delta_x) * (stencil3 - stencil4);
}

__device__ double duv_dy_device(double** u, double** v, int i, int j, double delta_y, double gamma) {
    double stencil1 = 0.5 * (v[i][j] + v[i+1][j]);
    double stencil2 = 0.5 * (v[i][j-1] + v[i+1][j-1]);

    double stencil3 = stencil1 * 0.5 * (u[i][j] + u[i][j+1]);
    double stencil4 = stencil2 * 0.5 * (u[i][j-1] + u[i][j]);

    double stencil5 = fabs(stencil1) * 0.5 * (u[i][j] - u[i][j+1]);
    double stencil6 = fabs(stencil2) * 0.5 * (u[i][j-1] - u[i][j]);

    return (1.0/delta_y) * (stencil3 - stencil4) + (gamma / delta_y) * (stencil5 - stencil6);
}

__device__ double dv2_dy_device(double** u, double** v, int i, int j, double delta_y, double gamma) {
    double stencil1 = 0.5 * (v[i][j] + v[i][j+1]);
    double stencil2 = 0.5 * (v[i][j-1] + v[i][j]);

    double stencil3 = fabs(stencil1) * 0.5 * (v[i][j] - v[i][j+1]);
    double stencil4 = fabs(stencil2) * 0.5 * (v[i][j-1] - v[i][j]);

    return (1.0/delta_y) * (stencil1*stencil1 - stencil2*stencil2) + (gamma / delta_y) * (stencil3 - stencil4);
}

__device__ double duv_dx_device(double** u, double** v, int i, int j, double delta_x, double gamma) {
    double stencil1 = 0.5 * (u[i][j] + u[i][j+1]);
    double stencil2 = 0.5 * (u[i-1][j] + u[i-1][j+1]);

    double stencil3 = stencil1 * 0.5 * (v[i][j] + v[i+1][j]);
    double stencil4 = stencil2 * 0.5 * (v[i-1][j] + v[i][j]);

    double stencil5 = fabs(stencil1) * 0.5 * (v[i][j] - v[i+1][j]);
    double stencil6 = fabs(stencil2) * 0.5 * (v[i-1][j] - v[i][j]);

    return (1.0/delta_x) * (stencil3 - stencil4) + (gamma / delta_x) * (stencil5 - stencil6);
}

// Central differences for second derivatives
__device__ double d2u_dx2_device(double** u, int i, int j, double delta_x) {
    return (u[i+1][j] - 2.0 * u[i][j] + u[i-1][j]) / (delta_x * delta_x);
}

__device__ double d2u_dy2_device(double** u, int i, int j, double delta_y) {
    return (u[i][j+1] - 2.0 * u[i][j] + u[i][j-1]) / (delta_y * delta_y);
}

__device__ double d2v_dx2_device(double** v, int i, int j, double delta_x) {
    return (v[i+1][j] - 2.0 * v[i][j] + v[i-1][j]) / (delta_x * delta_x);
}

__device__ double d2v_dy2_device(double** v, int i, int j, double delta_y) {
    return (v[i][j+1] - 2.0 * v[i][j] + v[i][j-1]) / (delta_y * delta_y);
}

__global__ void calculate_F_kernel(double **F, double **u, double **v, int i_max, int j_max, 
                                  double Re, double g_x, double delta_t, double delta_x, 
                                  double delta_y, double gamma) {
    int i = blockIdx.x * blockDim.x + threadIdx.x + 1;
    int j = blockIdx.y * blockDim.y + threadIdx.y + 1;
    
    if (i <= i_max-1 && j <= j_max) {  // F bounds: i from 1 to i_max-1
        F[i][j] = u[i][j] + delta_t * (
            (1.0/Re) * (d2u_dx2_device(u, i, j, delta_x) + d2u_dy2_device(u, i, j, delta_y)) 
            - du2_dx_device(u, v, i, j, delta_x, gamma) 
            - duv_dy_device(u, v, i, j, delta_y, gamma) 
            + g_x
        );
    }
}

__global__ void calculate_G_kernel(double **G, double **u, double **v, int i_max, int j_max, 
                                  double Re, double g_y, double delta_t, double delta_x, 
                                  double delta_y, double gamma) {
    int i = blockIdx.x * blockDim.x + threadIdx.x + 1;
    int j = blockIdx.y * blockDim.y + threadIdx.y + 1;
    
    if (i <= i_max && j <= j_max-1) {  // G bounds: j from 1 to j_max-1
        G[i][j] = v[i][j] + delta_t * (
            (1.0/Re) * (d2v_dx2_device(v, i, j, delta_x) + d2v_dy2_device(v, i, j, delta_y)) 
            - duv_dx_device(u, v, i, j, delta_x, gamma) 
            - dv2_dy_device(u, v, i, j, delta_y, gamma) 
            + g_y
        );
    }
}

__global__ void sor_shared_memory_kernel(double **p, double **RHS, 
                                        int i_max, int j_max, double delta_x, double delta_y, 
                                        double omega, int color, int block_size) {
    // Use dynamic shared memory instead of static arrays
    extern __shared__ double shared_mem[];
    
    const int SHARED_MEM_DIM_SIZE = block_size + 2;
    
    // Layout: p_shared first, then RHS_shared
    double *p_shared_data = shared_mem;
    double *RHS_shared_data = shared_mem + SHARED_MEM_DIM_SIZE * (SHARED_MEM_DIM_SIZE + 1);
    
    // Helper macros to access 2D arrays in 1D memory
    #define P_SHARED(i,j) p_shared_data[(i) * (SHARED_MEM_DIM_SIZE + 1) + (j)]
    #define RHS_SHARED(i,j) RHS_shared_data[(i) * block_size + (j)]

    int tx = threadIdx.x; // 0 to block_size-1
    int ty = threadIdx.y; // 0 to block_size-1

    // Global indices for the current thread's primary responsibility (center of its 3x3 stencil in shared mem)
    int current_i = blockIdx.x * block_size + tx + 1; // 1 to i_max
    int current_j = blockIdx.y * block_size + ty + 1; // 1 to j_max

    // --- Stage 1: Load data into shared memory ---

    // Each thread loads its corresponding p[current_i][current_j] into the center of its shared memory view
    // p_shared[tx+1][ty+1] corresponds to p[current_i][current_j]
    if (current_i >= 1 && current_i <= i_max && current_j >= 1 && current_j <= j_max) {
        P_SHARED(tx + 1, ty + 1) = p[current_i][current_j];
        RHS_SHARED(tx, ty) = RHS[current_i][current_j]; // RHS_shared is indexed 0..block_size-1
    }

    // Load halo regions into p_shared
    // Global indices for p array are 0 to i_max+1 and 0 to j_max+1

    // Left halo: p_shared[0][ty+1]
    if (tx == 0) {
        int gi = blockIdx.x * block_size; // Global i for the halo element p[gi][gj]
        int gj = blockIdx.y * block_size + ty + 1;
        if (gi >= 0 && gi <= i_max + 1 && gj >= 1 && gj <= j_max) { // Check gj bounds carefully
            P_SHARED(0, ty + 1) = p[gi][gj];
        }
    }
    // Right halo: p_shared[block_size+1][ty+1]
    if (tx == block_size - 1) {
        int gi = blockIdx.x * block_size + block_size + 1;
        int gj = blockIdx.y * block_size + ty + 1;
        if (gi >= 0 && gi <= i_max + 1 && gj >= 1 && gj <= j_max) {
            P_SHARED(block_size + 1, ty + 1) = p[gi][gj];
        }
    }
    // Top halo: p_shared[tx+1][0]
    if (ty == 0) {
        int gi = blockIdx.x * block_size + tx + 1;
        int gj = blockIdx.y * block_size;
        if (gi >= 1 && gi <= i_max && gj >= 0 && gj <= j_max + 1) { // Check gi bounds carefully
             P_SHARED(tx + 1, 0) = p[gi][gj];
        }
    }
    // Bottom halo: p_shared[tx+1][block_size+1]
    if (ty == block_size - 1) {
        int gi = blockIdx.x * block_size + tx + 1;
        int gj = blockIdx.y * block_size + block_size + 1;
        if (gi >= 1 && gi <= i_max && gj >= 0 && gj <= j_max + 1) {
            P_SHARED(tx + 1, block_size + 1) = p[gi][gj];
        }
    }

    // Corner halos for p_shared
    // Top-Left: p_shared[0][0]
    if (tx == 0 && ty == 0) {
        int gi = blockIdx.x * block_size;
        int gj = blockIdx.y * block_size;
        if (gi >= 0 && gi <= i_max + 1 && gj >= 0 && gj <= j_max + 1) {
            P_SHARED(0, 0) = p[gi][gj];
        }
    }
    // Top-Right: p_shared[block_size+1][0]
    if (tx == block_size - 1 && ty == 0) {
        int gi = blockIdx.x * block_size + block_size + 1;
        int gj = blockIdx.y * block_size;
        if (gi >= 0 && gi <= i_max + 1 && gj >= 0 && gj <= j_max + 1) {
            P_SHARED(block_size + 1, 0) = p[gi][gj];
        }
    }
    // Bottom-Left: p_shared[0][block_size+1]
    if (tx == 0 && ty == block_size - 1) {
        int gi = blockIdx.x * block_size;
        int gj = blockIdx.y * block_size + block_size + 1;
        if (gi >= 0 && gi <= i_max + 1 && gj >= 0 && gj <= j_max + 1) {
            P_SHARED(0, block_size + 1) = p[gi][gj];
        }
    }
    // Bottom-Right: p_shared[block_size+1][block_size+1]
    if (tx == block_size - 1 && ty == block_size - 1) {
        int gi = blockIdx.x * block_size + block_size + 1;
        int gj = blockIdx.y * block_size + block_size + 1;
        if (gi >= 0 && gi <= i_max + 1 && gj >= 0 && gj <= j_max + 1) {
            P_SHARED(block_size + 1, block_size + 1) = p[gi][gj];
        }
    }

    __syncthreads();

    // --- Stage 2: Perform SOR update using shared memory ---
    // Computation is for p[current_i][current_j]
    if (current_i >= 1 && current_i <= i_max && current_j >= 1 && current_j <= j_max && (current_i + current_j) % 2 == color) {
        double dx2 = delta_x * delta_x;
        double dy2 = delta_y * delta_y;
        double coeff = 2.0 * (1.0/dx2 + 1.0/dy2);
        
        // Access p_shared using indices relative to the current thread's (tx,ty)
        // Center: p_shared[tx+1][ty+1]
        // Left:   p_shared[tx  ][ty+1]
        // Right:  p_shared[tx+2][ty+1]
        // Top:    p_shared[tx+1][ty  ]
        // Bottom: p_shared[tx+1][ty+2]
        double p_old = P_SHARED(tx + 1, ty + 1);
        double p_new = (1.0 - omega) * p_old + 
                        omega / coeff * 
                        ((P_SHARED(tx + 2, ty + 1) + P_SHARED(tx, ty + 1)) / dx2 +
                        (P_SHARED(tx + 1, ty + 2) + P_SHARED(tx + 1, ty)) / dy2 -
                        RHS_SHARED(tx, ty)); // Use the loaded interior RHS_shared

            // we are writing to the global mem, may affect performance
            p[current_i][current_j] = p_new;
        }
}

// Após o kernel sor_shared_memory_kernel

__global__ void calculate_residual_and_norm_kernel(double **p, double **RHS, 
                                                 int i_max, int j_max, 
                                                 double delta_x, double delta_y,
                                                 double *block_norms, int block_size) {
    
    // Shared memory for residual reduction
    extern __shared__ double res_shared[];
    
    int tx = threadIdx.x;
    int ty = threadIdx.y;
    
    // Índices globais
    int i = blockIdx.x * block_size + tx + 1;
    int j = blockIdx.y * block_size + ty + 1;
    
    double dx2 = delta_x * delta_x;
    double dy2 = delta_y * delta_y;
    double res_squared = 0.0;
    
    // Calcular resíduo para cada ponto e armazenar em memória compartilhada
    if (i <= i_max && j <= j_max) {
        double residual = (p[i+1][j] - 2.0 * p[i][j] + p[i-1][j]) / dx2 +
                          (p[i][j+1] - 2.0 * p[i][j] + p[i][j-1]) / dy2 -
                          RHS[i][j];
        
        // Armazenar o quadrado do resíduo para posterior redução
        res_squared = residual * residual;
        res_shared[tx * block_size + ty] = res_squared;
    } else {
        res_shared[tx * block_size + ty] = 0.0;
    }
    
    __syncthreads();
    
    // Redução paralela dentro do bloco
    for (int stride = (block_size * block_size)/2; stride > 0; stride >>= 1) {
        if (tx * block_size + ty < stride) {
            res_shared[tx * block_size + ty] += res_shared[tx * block_size + ty + stride];
        }
        __syncthreads();
    }
    
    // Thread (0,0) salva o resultado final do bloco
    if (tx == 0 && ty == 0) {
        block_norms[blockIdx.y * gridDim.x + blockIdx.x] = res_shared[0];
    }
}

// Kernel para redução final de normas de blocos para um único valor
__global__ void reduce_block_norms_kernel(double *block_norms, int num_blocks, double *final_norm, int i_max, int j_max, int block_size) {
    extern __shared__ double shared_data[]; // Use dynamic shared memory
    
    int tid = threadIdx.x;
    
    // Carregar dados para memória compartilhada
    double sum = 0.0;
    for (int i = tid; i < num_blocks; i += blockDim.x) {
        sum += block_norms[i];
    }
    shared_data[tid] = sum;
    
    __syncthreads();
    
    // Redução paralela
    for (int stride = blockDim.x/2; stride > 0; stride >>= 1) {
        if (tid < stride) {
            shared_data[tid] += shared_data[tid + stride];
        }
        __syncthreads();
    }
    
    // Thread 0 escreve o resultado final
    if (tid == 0) {
        *final_norm = sqrt(shared_data[0] / (i_max * j_max)); // Agora i_max e j_max são parâmetros
    }
}


__global__ void calculate_norm_kernel(double **matrix, int i_max, int j_max, double *block_norms, int block_size) {
    extern __shared__ double norm_shared[];
    
    int tx = threadIdx.x;
    int ty = threadIdx.y;
    int i = blockIdx.x * block_size + tx + 1;
    int j = blockIdx.y * block_size + ty + 1;
    
    double val = 0.0;
    if (i <= i_max && j <= j_max) {
        val = matrix[i][j];
        norm_shared[tx * block_size + ty] = val * val;
    } else {
        norm_shared[tx * block_size + ty] = 0.0;
    }
    
    __syncthreads();
    
    // Reduction in a single dimension
    for (int stride = (block_size * block_size)/2; stride > 0; stride >>= 1) {
        if (tx * block_size + ty < stride) {
            norm_shared[tx * block_size + ty] += norm_shared[tx * block_size + ty + stride];
        }
        __syncthreads();
    }
    
    if (tx == 0 && ty == 0) {
        block_norms[blockIdx.y * gridDim.x + blockIdx.x] = norm_shared[0];
    }
}



double calculate_L2_norm_device(double **matrix, int i_max, int j_max, int block_size) {

    dim3 blockDim(block_size, block_size);
    dim3 gridDim((i_max + block_size - 1) / block_size,
                 (j_max + block_size - 1) / block_size);
    
    int total_blocks = gridDim.x * gridDim.y;
    
    double *d_block_norms, *d_final_norm;
    CHECK_CUDA_ERROR(hipMalloc(&d_block_norms, total_blocks * sizeof(double)));
    CHECK_CUDA_ERROR(hipMalloc(&d_final_norm, sizeof(double)));
    
    // Calculate partial norms
    calculate_norm_kernel<<<gridDim, blockDim, block_size * block_size * sizeof(double)>>>(matrix, i_max, j_max, d_block_norms, block_size);
    CHECK_CUDA_ERROR(hipDeviceSynchronize());
    
    // Reduce to final norm
    reduce_block_norms_kernel<<<1, block_size * block_size, block_size * block_size * sizeof(double)>>>(d_block_norms, total_blocks, d_final_norm, i_max, j_max, block_size);
    CHECK_CUDA_ERROR(hipDeviceSynchronize());
    
    double norm;
    CHECK_CUDA_ERROR(hipMemcpy(&norm, d_final_norm, sizeof(double), hipMemcpyDeviceToHost));
    
    hipFree(d_block_norms);
    hipFree(d_final_norm);
    
    return norm;
}


int SOR_UVA_with_shared_memory(double **p, int i_max, int j_max, double delta_x, double delta_y,
                               double **res, double **RHS, double omega, double epsilon, int max_it,
                               BoundaryPoint *borders, int border_count, int block_size) {
    
    dim3 blockDim(block_size, block_size);
    dim3 gridDim((i_max + block_size - 1) / block_size,
                 (j_max + block_size - 1) / block_size);
    
    dim3 boundaryBlockDim(block_size * block_size); 
    dim3 boundaryGridDim((border_count + boundaryBlockDim.x - 1) / boundaryBlockDim.x);
    
    // Número total de blocos para o cálculo da norma
    int total_blocks = gridDim.x * gridDim.y;
    
    // Alocação de memória para normas de blocos e norma final
    double *d_block_norms, *d_final_norm;
    CHECK_CUDA_ERROR(hipMalloc(&d_block_norms, total_blocks * sizeof(double)));
    CHECK_CUDA_ERROR(hipMalloc(&d_final_norm, sizeof(double)));
    
    double norm_p_initial = calculate_L2_norm_device(p, i_max, j_max, block_size);
    double current_L2_res_norm;
    
    // Calculate shared memory sizes
    size_t sor_shared_mem_size = (block_size + 2) * (block_size + 3) * sizeof(double) + // P_SHARED
                                block_size * block_size * sizeof(double); // RHS_SHARED
    size_t calc_res_shared_mem_size = block_size * block_size * sizeof(double);
    size_t reduce_shared_mem_size = block_size * block_size * sizeof(double);
    
    for (int it = 0; it < max_it; it++) {
        // Atualizar bordas
        update_boundaries_with_precalc_kernel<<<boundaryGridDim, boundaryBlockDim>>>(p, borders, border_count);
        CHECK_CUDA_ERROR(hipDeviceSynchronize());
        
        // Pontos vermelhos com memória compartilhada
        sor_shared_memory_kernel<<<gridDim, blockDim, sor_shared_mem_size>>>(p, RHS, i_max, j_max, delta_x, delta_y, omega, 0, block_size);
        CHECK_CUDA_ERROR(hipDeviceSynchronize());
        
        // Atualizar bordas novamente
        update_boundaries_with_precalc_kernel<<<boundaryGridDim, boundaryBlockDim>>>(p, borders, border_count);
        CHECK_CUDA_ERROR(hipDeviceSynchronize());
        
        // Pontos pretos com memória compartilhada
        sor_shared_memory_kernel<<<gridDim, blockDim, sor_shared_mem_size>>>(p, RHS, i_max, j_max, delta_x, delta_y, omega, 1, block_size);
        CHECK_CUDA_ERROR(hipDeviceSynchronize());
        
        // Verificação de convergência usando o novo método otimizado
        calculate_residual_and_norm_kernel<<<gridDim, blockDim, calc_res_shared_mem_size>>>(p, RHS, i_max, j_max, delta_x, delta_y, d_block_norms, block_size);
        CHECK_CUDA_ERROR(hipDeviceSynchronize());
        
        // Redução final das normas de blocos em um único valor
        reduce_block_norms_kernel<<<1, block_size * block_size, reduce_shared_mem_size>>>(d_block_norms, total_blocks, d_final_norm, i_max, j_max, block_size);
        CHECK_CUDA_ERROR(hipDeviceSynchronize());
        
        // Transferir apenas o valor final da norma para o host
        CHECK_CUDA_ERROR(hipMemcpy(&current_L2_res_norm, d_final_norm, sizeof(double), hipMemcpyDeviceToHost));
        
        // Verificar convergência
        if (current_L2_res_norm <= epsilon * (norm_p_initial + 1.5)) {
            // Liberar memória e retornar
            hipFree(d_block_norms);
            hipFree(d_final_norm);
            return it + 1;
        }
    }
    
    // Liberar memória
    hipFree(d_block_norms);
    hipFree(d_final_norm);
    
    return -1; // Não convergiu
}


__global__ void find_max_kernel(double **matrix, int i_max, int j_max, double *block_max, int block_size) {
    
    // Compartilhar valores máximos para blocos - usar para redução local
    extern __shared__ double max_shared[]; // Use dynamic shared memory
    
    int tx = threadIdx.x;
    int ty = threadIdx.y;
    
    // Índices globais
    int i = blockIdx.x * block_size + tx + 1;
    int j = blockIdx.y * block_size + ty + 1;
    
    // Inicializar com valor mínimo
    double local_max = -1e30; // Valor negativo grande
    
    // Pegar o valor da matriz se estiver dentro dos limites
    if (i <= i_max && j <= j_max) {
        local_max = matrix[i][j];
    }
    
    // Armazenar valor local em memória compartilhada
    max_shared[tx * block_size + ty] = local_max;
    
    __syncthreads();
    
    // Redução paralela dentro do bloco - encontrando o máximo
    for (int stride = (block_size * block_size)/2; stride > 0; stride >>= 1) {
        if (tx * block_size + ty < stride) {
            max_shared[tx * block_size + ty] = fmax(max_shared[tx * block_size + ty], max_shared[tx * block_size + ty + stride]);
        }
        __syncthreads();
    }
    
    // Thread (0,0) salva o resultado final do bloco
    if (tx == 0 && ty == 0) {
        block_max[blockIdx.y * gridDim.x + blockIdx.x] = max_shared[0];
    }
}

// Kernel para redução final dos máximos de blocos para um único valor
__global__ void reduce_block_max_kernel(double *block_max, int num_blocks, double *final_max, int block_size) {
    extern __shared__ double shared_data[];
    
    int tid = threadIdx.x;
    
    // Inicializar com valor mínimo
    shared_data[tid] = -1e30;
    
    // Carregar dados para memória compartilhada
    for (int i = tid; i < num_blocks; i += blockDim.x) {
        shared_data[tid] = fmax(shared_data[tid], block_max[i]);
    }
    
    __syncthreads();
    
    // Redução paralela para encontrar o máximo global
    for (int stride = blockDim.x/2; stride > 0; stride >>= 1) {
        if (tid < stride) {
            shared_data[tid] = fmax(shared_data[tid], shared_data[tid + stride]);
        }
        __syncthreads();
    }
    
    // Thread 0 escreve o resultado final
    if (tid == 0) {
        *final_max = shared_data[0];
    }
}

// Função host para encontrar o máximo de uma matriz usando CUDA
double max_mat_cuda(int i_max, int j_max, double **matrix) {
    const int BLOCK_SIZE = 16; // Default block size for this function
    
    // Configurar dimensões da grade e blocos
    dim3 blockDim(BLOCK_SIZE, BLOCK_SIZE);
    dim3 gridDim((i_max + BLOCK_SIZE - 1) / BLOCK_SIZE,
                 (j_max + BLOCK_SIZE - 1) / BLOCK_SIZE);
    
    // Número total de blocos
    int total_blocks = gridDim.x * gridDim.y;
    
    // Alocar memória para máximos de blocos e máximo final
    double *d_block_max, *d_final_max;
    CHECK_CUDA_ERROR(hipMalloc(&d_block_max, total_blocks * sizeof(double)));
    CHECK_CUDA_ERROR(hipMalloc(&d_final_max, sizeof(double)));
    
    // Encontrar máximos locais em cada bloco
    find_max_kernel<<<gridDim, blockDim, BLOCK_SIZE * BLOCK_SIZE * sizeof(double)>>>(matrix, i_max, j_max, d_block_max, BLOCK_SIZE);
    CHECK_CUDA_ERROR(hipDeviceSynchronize());
    
    // Reduzir máximos locais para um máximo global
    reduce_block_max_kernel<<<1, BLOCK_SIZE * BLOCK_SIZE, BLOCK_SIZE * BLOCK_SIZE * sizeof(double)>>>(d_block_max, total_blocks, d_final_max, BLOCK_SIZE);
    CHECK_CUDA_ERROR(hipDeviceSynchronize());
    
    // Transferir o máximo final para o host
    double max_value;
    CHECK_CUDA_ERROR(hipMemcpy(&max_value, d_final_max, sizeof(double), hipMemcpyDeviceToHost));
    
    // Liberar memória
    hipFree(d_block_max);
    hipFree(d_final_max);
    
    return max_value;
}


// Adicione após a definição de BoundaryPoint no início do arquivo

// Kernel para aplicar condições de contorno de não-deslizamento (no-slip)
__global__ void set_noslip_kernel(double **u, double **v, BoundaryPoint *borders, 
                                  int border_count, int side) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    
    if (idx < border_count) {
        // Verifica se este ponto de borda corresponde ao lado desejado
        if (borders[idx].position == side) {
            int i = borders[idx].i;
            int j = borders[idx].j;
            
            // Aplica condições de não-deslizamento conforme o lado
            switch (side) {
                case TOP:
                    if (j == borders[idx].j) { // Confirma que estamos na borda superior
                        v[i][j] = 0.0; // Velocidade v fixa na borda
                        u[i][j] = -u[i][j-1]; // Reflexão da velocidade u
                    }
                    break;
                    
                case BOTTOM:
                    if (j == borders[idx].j) { // Confirma que estamos na borda inferior
                        v[i][j] = 0.0; // Velocidade v fixa na borda
                        u[i][j] = -u[i][j+1]; // Reflexão da velocidade u
                    }
                    break;
                    
                case LEFT:
                    if (i == borders[idx].i) { // Confirma que estamos na borda esquerda
                        u[i][j] = 0.0; // Velocidade u fixa na borda
                        v[i][j] = -v[i+1][j]; // Reflexão da velocidade v
                    }
                    break;
                    
                case RIGHT:
                    if (i == borders[idx].i) { // Confirma que estamos na borda direita
                        u[i][j] = 0.0; // Velocidade u fixa na borda
                        v[i][j] = -v[i-1][j]; // Reflexão da velocidade v
                    }
                    break;
            }
        }
    }
}

// Kernel para aplicar condições de contorno de entrada (inflow)
__global__ void set_inflow_kernel(double **u, double **v, BoundaryPoint *borders, 
                                 int border_count, int side, double u_fix, double v_fix) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    
    if (idx < border_count) {
        // Verifica se este ponto de borda corresponde ao lado desejado
        if (borders[idx].position == side) {
            int i = borders[idx].i;
            int j = borders[idx].j;
            
            // Aplica condições de entrada conforme o lado
            switch (side) {
                case TOP:
                    if (j == borders[idx].j) { // Confirma que estamos na borda superior
                        v[i][j] = v_fix; // Velocidade v fixa na borda
                        u[i][j] = 2 * u_fix - u[i][j-1]; // Valor extrapolado para u
                    }
                    break;
                    
                case BOTTOM:
                    if (j == borders[idx].j) { // Confirma que estamos na borda inferior
                        v[i][j] = v_fix; // Velocidade v fixa na borda
                        u[i][j] = 2 * u_fix - u[i][j+1]; // Valor extrapolado para u
                    }
                    break;
                    
                case LEFT:
                    if (i == borders[idx].i) { // Confirma que estamos na borda esquerda
                        u[i][j] = u_fix; // Velocidade u fixa na borda
                        v[i][j] = 2 * v_fix - v[i+1][j]; // Valor extrapolado para v
                    }
                    break;
                    
                case RIGHT:
                    if (i == borders[idx].i) { // Confirma que estamos na borda direita
                        u[i][j] = u_fix; // Velocidade u fixa na borda
                        v[i][j] = 2 * v_fix - v[i-1][j]; // Valor extrapolado para v
                    }
                    break;
            }
        }
    }
}

// Funções host para invocar os kernels
void set_noslip_cuda(int i_max, int j_max, double **u, double **v, int side, 
                     BoundaryPoint *borders, int border_count) {
    dim3 blockDim(BLOCK_SIZE * BLOCK_SIZE);
    dim3 gridDim((border_count + blockDim.x - 1) / blockDim.x);
    
    set_noslip_kernel<<<gridDim, blockDim>>>(u, v, borders, border_count, side);
    CHECK_CUDA_ERROR(hipDeviceSynchronize());
}

void set_inflow_cuda(int i_max, int j_max, double **u, double **v, int side, 
                     double u_fix, double v_fix, BoundaryPoint *borders, int border_count) {
    dim3 blockDim(BLOCK_SIZE * BLOCK_SIZE);
    dim3 gridDim((border_count + blockDim.x - 1) / blockDim.x);
    
    set_inflow_kernel<<<gridDim, blockDim>>>(u, v, borders, border_count, side, u_fix, v_fix);
    CHECK_CUDA_ERROR(hipDeviceSynchronize());
}


/**
 * @brief Main function.
 * 
 * This is the main function.
 * @return 0 on exit.
 */

int main(int argc, char* argv[])
{
    // Grid pointers - agora serão alocados com UVA
    double** u;     // velocity x-component
    double** v;     // velocity y-component
    double** p;     // pressure
    double** F;     // F term
    double** G;     // G term
    double** res;   // SOR residuum
    double** RHS;   // RHS of poisson equation
    BoundaryPoint* borders; // Array to store border points
    // Simulation parameters.
    int i_max, j_max;                   // number of grid points in each direction
    double a, b;                        // sizes of the grid
    double Re;                          // reynolds number
    double delta_t, delta_x, delta_y;   // step sizes
    double gamma;                       // weight for Donor-Cell-stencil
    double T;                           // max time for integration
    double g_x;                         // x-component of g
    double g_y;                         // y-component of g
    double tau;                         // security factor for adaptive step size
    double omega;                       // relaxation parameter
    double epsilon;                     // relative tolerance for SOR
    int max_it;                         // maximum iterations for SOR
    int n_print;                        // output to file every ..th step
    int problem;                        // problem type
    double f;                           // frequency of periodic boundary conditions (if problem == 2)

    const char* param_file = "parameters.txt"; // file containing parameters

    // fprintf(stderr, "CUDA: Working directory test\n");
    
    // Check if block size is provided as second argument
    if (argc > 2) {
        int new_block_size = atoi(argv[2]);
        if (new_block_size > 0 && new_block_size <= 32) { // Reasonable range check
            BLOCK_SIZE = new_block_size;
        } else {
            fprintf(stderr, "Warning: Invalid block size %d. Using default %d\n", new_block_size, BLOCK_SIZE);
        }
    }
    if(BLOCK_SIZE == -1){
        fprintf(stderr, "Error: Block size not specified. Please provide a valid block size as the second argument.\n");
        return 1;
    } else {
        fprintf(stderr, "CUDA: Using block size %d\n", BLOCK_SIZE);
    }
    
    // Test if we can open the file directly
    if (argc > 1) {
        FILE *fp = fopen(argv[1], "r");
        if (fp == NULL) {
            fprintf(stderr, "CUDA: Could not open param_file\n");
        } else {
            // fprintf(stderr, "CUDA: Successfully opened '%s'\n", argv[1]);
            param_file = argv[1];
            fclose(fp);
        }
    }
    
    // if (argc > 1) {
    //     param_file = argv[1];
    // }
    
    // Initialize all parameters.
    init(&problem, &f, &i_max, &j_max, &a, &b, &Re, &T, &g_x, &g_y, &tau, &omega, &epsilon, &max_it, &n_print, param_file);
    // Set step size in space.
    delta_x = a / i_max;
    delta_y = b / j_max;

    // Calcular o número exato de pontos de borda
    // A fórmula é 2 * ( (i_max+2) + (j_max+2) - 2 ) = 2 * (i_max + j_max + 2)
    // (soma dos comprimentos das bordas, subtraindo os 4 cantos contados duas vezes, mas cada célula de canto é um ponto)
    // Ou mais simples: (i_max+2)*2 para bordas superior/inferior + j_max*2 para bordas laterais (excluindo cantos já contados)
    // = 2*i_max + 4 + 2*j_max = 2 * (i_max + j_max + 2)
    int num_actual_border_points = 2 * (i_max + j_max + 2);

    // Passar num_actual_border_points para allocate_unified_memory
    allocate_device_memory(&u, &v, &p, &res, &RHS, &F, &G, i_max, j_max, &borders, num_actual_border_points);    // precalculate_borders preenche o array 'borders'.
    // Ele não precisa mais do count como parâmetro se a memória já está dimensionada corretamente.
    precalculate_borders(i_max, j_max, borders);
    
    // Allocate memory using UVA instead of regular allocation

    // Time loop.
    double t = 0;
    int i, j;
    int n = 0;
    int n_out = 0;

    clock_t start = clock();
    double time_sor = 0.0;
    while (t < T) {
        // Adaptive stepsize and weight factor for Donor-Cell
        double u_max = max_mat_cuda(i_max, j_max, u);
        double v_max = max_mat_cuda(i_max, j_max, v);
        delta_t = tau * n_min(3, Re / 2.0 / ( 1.0 / delta_x / delta_x + 1.0 / delta_y / delta_y ), delta_x / fabs(u_max), delta_y / fabs(v_max));
        gamma = fmax(u_max * delta_t / delta_x, v_max * delta_t / delta_y);

        // Set boundary conditions (permanecem na CPU)
        if (problem == 1) {
            set_noslip_cuda(i_max, j_max, u, v, LEFT, borders, num_actual_border_points);
            set_noslip_cuda(i_max, j_max, u, v, RIGHT, borders, num_actual_border_points);
            set_noslip_cuda(i_max, j_max, u, v, BOTTOM, borders, num_actual_border_points);
            set_inflow_cuda(i_max, j_max, u, v, TOP, 1.0, 0.0, borders, num_actual_border_points);
        } else if (problem == 2) {
            set_noslip_cuda(i_max, j_max, u, v, LEFT, borders, num_actual_border_points);
            set_noslip_cuda(i_max, j_max, u, v, RIGHT, borders, num_actual_border_points);
            set_noslip_cuda(i_max, j_max, u, v, BOTTOM, borders, num_actual_border_points);
            set_inflow_cuda(i_max, j_max, u, v, TOP, sin(f*t), 0.0, borders, num_actual_border_points);           
        }

        dim3 blockDim(16, 16);
        dim3 gridDim((i_max + blockDim.x - 1) / blockDim.x,
                     (j_max + blockDim.y - 1) / blockDim.y);

        // Calculate F and G (pode ser mantido na CPU ou implementado em CUDA)
        calculate_F_kernel<<<gridDim, blockDim>>>(F, u, v, i_max, j_max, Re, g_x, delta_t, delta_x, delta_y, gamma);
        CHECK_CUDA_ERROR(hipGetLastError());

        calculate_G_kernel<<<gridDim, blockDim>>>(G, u, v, i_max, j_max, Re, g_y, delta_t, delta_x, delta_y, gamma);
        CHECK_CUDA_ERROR(hipGetLastError());
        CHECK_CUDA_ERROR(hipDeviceSynchronize());

        // RHS of Poisson equation - now using CUDA kernel
        calculate_RHS_kernel<<<gridDim, blockDim>>>(RHS, F, G, i_max, j_max, delta_t, delta_x, delta_y);
        CHECK_CUDA_ERROR(hipGetLastError());
        CHECK_CUDA_ERROR(hipDeviceSynchronize());
        //clock_t start_sor = clock();
        // Execute SOR step using UVA
        // Passar num_actual_border_points para SOR_UVA
        //SOR_UVA(p, i_max, j_max, delta_x, delta_y, res, RHS, omega, epsilon, max_it, borders, num_actual_border_points);
        clock_t start_sor = clock();
        SOR_UVA_with_shared_memory(p, i_max, j_max, delta_x, delta_y, res, RHS, omega, epsilon, max_it, borders, num_actual_border_points, BLOCK_SIZE);
        clock_t end_sor = clock();
        time_sor += (double)(end_sor - start_sor) / CLOCKS_PER_SEC;

        CHECK_CUDA_ERROR(hipGetLastError());
        //clock_t end_sor = clock();
        //double sor_time = (double)(end_sor - start_sor) / CLOCKS_PER_SEC;
        //fprintf(stderr, "SOR time: %.6f\n", sor_time);

        // Update velocities using CUDA kernel
        update_velocities_kernel<<<gridDim, blockDim>>>(u, v, F, G, p, i_max, j_max, delta_t, delta_x, delta_y);
        CHECK_CUDA_ERROR(hipGetLastError());
        CHECK_CUDA_ERROR(hipDeviceSynchronize());

        t += delta_t;
        n++;
    }

    // Get center values from device memory
    double u_center, v_center;
    double *u_row_ptr, *v_row_ptr;
    int center_i = i_max/2;
    int center_j = j_max/2;

    // First get the row pointers
    CHECK_CUDA_ERROR(hipMemcpy(&u_row_ptr, u + center_i, sizeof(double*), hipMemcpyDeviceToHost));
    CHECK_CUDA_ERROR(hipMemcpy(&v_row_ptr, v + center_i, sizeof(double*), hipMemcpyDeviceToHost));

    // Then get the actual values
    CHECK_CUDA_ERROR(hipMemcpy(&u_center, u_row_ptr + center_j, sizeof(double), hipMemcpyDeviceToHost));
    CHECK_CUDA_ERROR(hipMemcpy(&v_center, v_row_ptr + center_j, sizeof(double), hipMemcpyDeviceToHost));

    printf("U-CENTER: %.6f\n", u_center);
    printf("V-CENTER: %.6f\n", v_center);

    clock_t end = clock();
    double time_spent = (double)(end - start) / CLOCKS_PER_SEC;
    fprintf(stderr, "%.6f", time_sor);

    // Free unified memory
    free_device_memory(u, v, p, res, RHS, F, G, borders);
    return 0;
}

