#include "hip/hip_runtime.h"
#include "memory.h"
#include "io.h"
#include "integration.h"
#include "boundaries.h"
#include "cuda_kernels.h"
#include "utils.h"

#include <time.h>
#include <math.h>
#include <stdio.h>

int main(int argc, char* argv[])
{
    double** u;     
    double** v;     
    double** p;     

    double** F;     
    double** G;     
    double** res;   
    double** RHS;   

    int i_max, j_max;                   
    double a, b;                        
    double Re;                          
    double delta_t, delta_x, delta_y;   
    double gamma;                       
    double T;                           
    double g_x;                         
    double g_y;                         
    double tau;                         
    double omega;                       
    double epsilon;                     
    int max_it;                         
    int n_print;                        
    int problem;                        
    double f;                           

    const char* param_file = "parameters.txt"; 

    if (argc > 1) {
        FILE *fp = fopen(argv[1], "r");
        if (fp == NULL) {
            fprintf(stderr, "CUDA: Could not open param_file\n");
        } else {
            param_file = argv[1];
            fclose(fp);
        }
    }
    
    init(&problem, &f, &i_max, &j_max, &a, &b, &T, &Re, &g_x, &g_y, &tau, &omega, &epsilon, &max_it, &n_print, param_file);
    printf("Initialized!\n");

    delta_x = a / i_max;
    delta_y = b / j_max;

    allocate_memory(&u, &v, &p, &res, &RHS, &F, &G, i_max, j_max);
    printf("Memory allocated.\n");
    
    // Inicialização dos arrays CUDA
    initCudaArrays(i_max, j_max);
    printf("CUDA arrays initialized.\n");

    double t = 0;
    int i, j;
    int n = 0;
    int n_out = 0;

    clock_t start = clock();

    while (t < T) {
        if (n % n_print == 0) {
            printf("%.5f / %.5f\n---------------------\n", t, T);
        }
        
        int sor_result = cudaSOR(p, u, v, i_max, j_max, delta_x, delta_y, res, RHS, omega, epsilon, max_it, F, G, tau,
                                 Re, problem, f, &t, &n_out);
        
        // t já é atualizado dentro de cudaSOR
        n++;
    }
    
    clock_t end = clock();
    double time_spent = (double)(end - start) / CLOCKS_PER_SEC;
    fprintf(stderr, "%.6f", time_spent);
    
    // Liberar memória CUDA
    freeCudaArrays();
    
    // Liberar memória CPU
    free_memory(&u, &v, &p, &res, &RHS, &F, &G, i_max);
    return 0;
}