#include "hip/hip_runtime.h"
#include "cuda_kernels.h"
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <math.h>
#include <stdarg.h>

// Funções utilitárias para CUDA

// Debug level: 0-none, 1-errors, 2-warnings, 3-info, 4-verbose
#define DEBUG_LEVEL 0

// Macros for debug printing
#define DEBUG_ERROR(fmt, ...) if (DEBUG_LEVEL >= 1) { fprintf(stderr, "ERROR: " fmt "\n", ##__VA_ARGS__); }
#define DEBUG_WARN(fmt, ...)  if (DEBUG_LEVEL >= 2) { fprintf(stderr, "WARNING: " fmt "\n", ##__VA_ARGS__); }
#define DEBUG_INFO(fmt, ...)  if (DEBUG_LEVEL >= 3) { fprintf(stderr, "INFO: " fmt "\n", ##__VA_ARGS__); }
#define DEBUG_VERBOSE(fmt, ...) if (DEBUG_LEVEL >= 4) { fprintf(stderr, "DEBUG: " fmt "\n", ##__VA_ARGS__); }

// Enhanced CUDA error checking with context
void check_cuda(hipError_t error, const char *filename, const int line, const char *funcname = "")
{
  if (error != hipSuccess) {
    fprintf(stderr, "CUDA ERROR: %s:%d:%s: %s (%d: %s)\n", 
            filename, line, funcname,
            hipGetErrorName(error), error, hipGetErrorString(error));
    exit(EXIT_FAILURE);
  }
}

// Check for kernel launch errors
void check_kernel_launch(const char *kernel_name, const char *filename, const int line)
{
    hipError_t error = hipGetLastError();
    if (error != hipSuccess) {
        fprintf(stderr, "KERNEL LAUNCH ERROR (%s): %s:%d: %s (%s)\n", 
                kernel_name, filename, line,
                hipGetErrorName(error), hipGetErrorString(error));
        exit(EXIT_FAILURE);
    }
    
    error = hipDeviceSynchronize();
    if (error != hipSuccess) {
        fprintf(stderr, "KERNEL EXECUTION ERROR (%s): %s:%d: %s (%s)\n", 
                kernel_name, filename, line,
                hipGetErrorName(error), hipGetErrorString(error));
        exit(EXIT_FAILURE);
    }
}

// Memory bounds checking helper
bool check_mem_bounds(const void* ptr, size_t size, const char* ptr_name, const char* filename, int line) {
    if (ptr == NULL) {
        fprintf(stderr, "NULL POINTER: %s at %s:%d\n", ptr_name, filename, line);
        return false;
    }
    
    // We can't actually check bounds on GPU memory from host code in a portable way
    // This is just a placeholder for the NULL check
    return true;
}

#define CUDACHECK(cmd) check_cuda(cmd, __FILE__, __LINE__, __func__)
#define KERNEL_CHECK(kernel_name) check_kernel_launch(kernel_name, __FILE__, __LINE__)
#define CHECK_POINTER(ptr, size, name) check_mem_bounds((ptr), (size), (name), __FILE__, __LINE__)

// Variáveis globais para arrays unificados
double *unified_p = NULL;
double *unified_res = NULL;
double *unified_RHS = NULL;
double *unified_u = NULL;
double *unified_v = NULL;
double *unified_F = NULL;
double *unified_G = NULL;
double *unified_dpdx = NULL;
double *unified_dpdy = NULL;
double u_max = 0.0;
double v_max = 0.0;
double delta_t, delta_x, delta_y, gamma_factor;  // Alterado de gamma para gamma_factor
size_t cuda_array_size = 0;
int grid_i_max = 0;
int grid_j_max = 0;

__global__ void RedSORKernel(double* p, double* RHS, int i_max, int j_max, double omega, double dxdx, double dydy) {
    int i = blockIdx.x * blockDim.x + threadIdx.x + 1; // +1 to skip ghost cells
    int j = blockIdx.y * blockDim.y + threadIdx.y + 1; // +1 to skip ghost cells

    // Only update red cells (i+j is even)
    if (i <= i_max && j <= j_max && (i + j) % 2 == 0) {
        p[i * (j_max + 2) + j] = (1.0 - omega) * p[i * (j_max + 2) + j] + 
            omega / (2.0 * (1.0 / dxdx + 1.0 / dydy)) *
            ((p[(i + 1) * (j_max + 2) + j] + p[(i - 1) * (j_max + 2) + j]) / dxdx + 
            (p[i * (j_max + 2) + (j + 1)] + p[i * (j_max + 2) + (j - 1)]) / dydy - RHS[i * (j_max + 2) + j]);
    }
}

__global__ void BlackSORKernel(double* p, double* RHS, int i_max, int j_max, double omega, double dxdx, double dydy) {
    int i = blockIdx.x * blockDim.x + threadIdx.x + 1;
    int j = blockIdx.y * blockDim.y + threadIdx.y + 1;

    // Only update black cells (i+j is odd)
    if (i <= i_max && j <= j_max && (i + j) % 2 == 1) {
        p[i * (j_max + 2) + j] = (1.0 - omega) * p[i * (j_max + 2) + j] + 
            omega / (2.0 * (1.0 / dxdx + 1.0 / dydy)) *
            ((p[(i + 1) * (j_max + 2) + j] + p[(i - 1) * (j_max + 2) + j]) / dxdx + 
            (p[i * (j_max + 2) + (j + 1)] + p[i * (j_max + 2) + (j - 1)]) / dydy - RHS[i * (j_max + 2) + j]);
    }
}

__global__ void CalculateResidualKernel(double* p, double* res, double* RHS, int i_max, int j_max, double dxdx, double dydy)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x + 1;
    int j = blockIdx.y * blockDim.y + threadIdx.y + 1;

    if (i <= i_max && j <= j_max) {
        res[i * (j_max + 2) + j] = (p[(i + 1) * (j_max + 2) + j] - 2.0 * p[i * (j_max + 2) + j] + p[(i - 1) * (j_max + 2) + j]) / dxdx + 
            (p[i * (j_max + 2) + (j + 1)] - 2.0 * p[i * (j_max + 2) + j] + p[i * (j_max + 2) + (j - 1)]) / dydy - RHS[i * (j_max + 2) + j];
    }
    
}

// Initialize CUDA arrays once
int initCudaArrays(double** p, double** u, double** v, double** res, double** RHS, int i_max, int j_max) {
    // Verificar argumentos
    if (p == NULL || u == NULL || v == NULL || res == NULL || RHS == NULL) {
        return -1;
    }
    
    if (i_max <= 0 || j_max <= 0) {
        return -1;
    }
    
    // Salvar dimensões da grade
    grid_i_max = i_max;
    grid_j_max = j_max;
    
    // Calcular tamanho necessário
    cuda_array_size = (i_max + 2) * (j_max + 2) * sizeof(double);
    
    // Verificar se o tamanho calculado não é muito grande
    size_t free_mem = 0, total_mem = 0;
    CUDACHECK(hipMemGetInfo(&free_mem, &total_mem));
    
    size_t total_needed = 9 * cuda_array_size; // 9 arrays
    if (total_needed > free_mem) {
        return -1;
    }
    
    // Alocar memória unificada
    hipError_t err;
    
    err = hipMallocManaged(&unified_p, cuda_array_size);
    if (err != hipSuccess) {
        return -1;
    }
    
    err = hipMallocManaged(&unified_res, cuda_array_size);
    if (err != hipSuccess) {
        if (unified_p) hipFree(unified_p);
        return -1;
    }
    
    err = hipMallocManaged(&unified_RHS, cuda_array_size);
    if (err != hipSuccess) {
        if (unified_p) hipFree(unified_p);
        if (unified_res) hipFree(unified_res);
        return -1;
    }
    
    err = hipMallocManaged(&unified_u, cuda_array_size);
    if (err != hipSuccess) {
        if (unified_p) hipFree(unified_p);
        if (unified_res) hipFree(unified_res);
        if (unified_RHS) hipFree(unified_RHS);
        return -1;
    }
    
    err = hipMallocManaged(&unified_v, cuda_array_size);
    if (err != hipSuccess) {
        if (unified_p) hipFree(unified_p);
        if (unified_res) hipFree(unified_res);
        if (unified_RHS) hipFree(unified_RHS);
        if (unified_u) hipFree(unified_u);
        return -1;
    }
    
    err = hipMallocManaged(&unified_F, cuda_array_size);
    if (err != hipSuccess) {
        freeCudaArrays(); // Liberar o que já foi alocado
        return -1;
    }
    
    err = hipMallocManaged(&unified_G, cuda_array_size);
    if (err != hipSuccess) {
        freeCudaArrays(); // Liberar o que já foi alocado
        return -1;
    }
    
    err = hipMallocManaged(&unified_dpdx, cuda_array_size);
    if (err != hipSuccess) {
        freeCudaArrays(); // Liberar o que já foi alocado
        return -1;
    }
    
    err = hipMallocManaged(&unified_dpdy, cuda_array_size);
    if (err != hipSuccess) {
        freeCudaArrays(); // Liberar o que já foi alocado
        return -1;
    }
    
    // Verificar se as alocações foram bem-sucedidas
    if (!CHECK_POINTER(unified_p, cuda_array_size, "unified_p") ||
        !CHECK_POINTER(unified_res, cuda_array_size, "unified_res") ||
        !CHECK_POINTER(unified_RHS, cuda_array_size, "unified_RHS") ||
        !CHECK_POINTER(unified_u, cuda_array_size, "unified_u") ||
        !CHECK_POINTER(unified_v, cuda_array_size, "unified_v") ||
        !CHECK_POINTER(unified_F, cuda_array_size, "unified_F") ||
        !CHECK_POINTER(unified_G, cuda_array_size, "unified_G") ||
        !CHECK_POINTER(unified_dpdx, cuda_array_size, "unified_dpdx") ||
        !CHECK_POINTER(unified_dpdy, cuda_array_size, "unified_dpdy")) {
        freeCudaArrays();
        return -1;
    }
    
    // Inicializar os arrays com dados da CPU - ADICIONAR VERIFICAÇÃO DE PONTEIROS NULOS
    for (int i = 0; i <= i_max + 1; i++) {
        for (int j = 0; j <= j_max + 1; j++) {
            // Verificar se os ponteiros p[i], res[i], etc são válidos
            if (p[i] == NULL || res[i] == NULL || RHS[i] == NULL) {
                return -1;
            }
            
            // Verificação especial para u que tem dimensão i_max+1
            if (i <= i_max && u[i] == NULL) {
                return -1;
            }
            
            // Verificação especial para v que tem dimensão j_max+1
            if (v[i] == NULL) {
                return -1;
            }
            
            // Verificar se o índice linearizado está dentro dos limites
            size_t idx = i * (j_max + 2) + j;
            if (idx >= (i_max + 2) * (j_max + 2)) {
                return -1;
            }
            
            try {
                unified_p[idx] = p[i][j];
                unified_res[idx] = res[i][j];
                unified_RHS[idx] = RHS[i][j];
                
                // Verificações especiais para u e v devido às dimensões diferentes
                if (i <= i_max && j <= j_max + 1) {
                    unified_u[idx] = u[i][j];
                }
                
                if (i <= i_max + 1 && j <= j_max) {
                    unified_v[idx] = v[i][j];
                }
                
                unified_F[idx] = 0.0;
                unified_G[idx] = 0.0;
                unified_dpdx[idx] = 0.0;
                unified_dpdy[idx] = 0.0;
            } catch (...) {
                return -1;
            }
        }
    }
    
    // Prefetch para GPU
    int device = -1;
    CUDACHECK(hipGetDevice(&device));
    
    CUDACHECK(hipMemPrefetchAsync(unified_p, cuda_array_size, device, NULL));
    CUDACHECK(hipMemPrefetchAsync(unified_RHS, cuda_array_size, device, NULL));
    CUDACHECK(hipMemPrefetchAsync(unified_res, cuda_array_size, device, NULL));
    CUDACHECK(hipMemPrefetchAsync(unified_u, cuda_array_size, device, NULL));
    CUDACHECK(hipMemPrefetchAsync(unified_v, cuda_array_size, device, NULL));
    CUDACHECK(hipMemPrefetchAsync(unified_F, cuda_array_size, device, NULL));
    CUDACHECK(hipMemPrefetchAsync(unified_G, cuda_array_size, device, NULL));
    CUDACHECK(hipMemPrefetchAsync(unified_dpdx, cuda_array_size, device, NULL));
    CUDACHECK(hipMemPrefetchAsync(unified_dpdy, cuda_array_size, device, NULL));
    
    return 0;
}

// Free CUDA arrays once at the end
void freeCudaArrays() {
    if (unified_p) {
        CUDACHECK(hipFree(unified_p));
        unified_p = NULL;
    }
    
    if (unified_res) {
        CUDACHECK(hipFree(unified_res));
        unified_res = NULL;
    }
    
    if (unified_RHS) {
        CUDACHECK(hipFree(unified_RHS));
        unified_RHS = NULL;
    }
    
    if (unified_u) {
        CUDACHECK(hipFree(unified_u));
        unified_u = NULL;
    }
    
    if (unified_v) {
        CUDACHECK(hipFree(unified_v));
        unified_v = NULL;
    }
    
    if (unified_F) {
        CUDACHECK(hipFree(unified_F));
        unified_F = NULL;
    }
    
    if (unified_G) {
        CUDACHECK(hipFree(unified_G));
        unified_G = NULL;
    }
    
    if (unified_dpdx) {
        CUDACHECK(hipFree(unified_dpdx));
        unified_dpdx = NULL;
    }
    
    if (unified_dpdy) {
        CUDACHECK(hipFree(unified_dpdy));
        unified_dpdy = NULL;
    }
}

int cudaSOR(double** p,double** u,double** v, int i_max, int j_max, double delta_x, double delta_y, 
            double** res, double** RHS, double omega, double eps, int max_it, double** F, double** G, double tau, double Re,
            int problem, double f, double* t, int* n_out, double g_x, double g_y) {
    int it = 0;
    double dydy = delta_y * delta_y;
    double dxdx = delta_x * delta_x;
    double norm_p = 0.0;
    
    // Calcular norma inicial de pressão
    for (int i = 1; i <= i_max; i++) {
        for(int j = 1; j <= j_max; j++) {
            norm_p += unified_p[i * (j_max + 2) + j] * unified_p[i * (j_max + 2) + j];
        }
    }
    norm_p = sqrt(norm_p / i_max / j_max);
    
    // 1. Calcular u_max e v_max usando kernel max_mat_kernel
    double *d_umax, *d_vmax;
    CUDACHECK(hipMallocManaged(&d_umax, sizeof(double)));
    CUDACHECK(hipMallocManaged(&d_vmax, sizeof(double)));
    
    if (!CHECK_POINTER(d_umax, sizeof(double), "d_umax") || 
        !CHECK_POINTER(d_vmax, sizeof(double), "d_vmax")) {
        return -1;
    }
    
    *d_umax = 0.0;
    *d_vmax = 0.0;
    int max_blocks = 32; // Ajuste conforme o tamanho do domínio
    int max_threads = 256;
    
    max_mat_kernel_double<<<max_blocks, max_threads>>>(unified_u, i_max, j_max, d_umax);
    KERNEL_CHECK("max_mat_kernel_double (u)");
    
    max_mat_kernel_double<<<max_blocks, max_threads>>>(unified_v, i_max, j_max, d_vmax);
    KERNEL_CHECK("max_mat_kernel_double (v)");
    
    u_max = *d_umax;
    v_max = *d_vmax;
    
    
    CUDACHECK(hipFree(d_umax));
    CUDACHECK(hipFree(d_vmax));
    
    // Calcular delta_t e gamma_factor
    delta_t = tau * n_min(4, 3.0, Re / 2.0 / ( 1.0 / delta_x / delta_x + 1.0 / delta_y / delta_y ), delta_x / fabs(u_max), delta_y / fabs(v_max));
    gamma_factor = fmax(u_max * delta_t / delta_x, v_max * delta_t / delta_y);
    
    
    // 1. Boundary conditions (GPU)
    dim3 block1D_j((j_max + 127) / 128); // for sides with j_max
    dim3 block1D_i((i_max + 127) / 128); // for sides with i_max
    int threads1D = 128;
    if (problem == 1){
        set_noslip_linear_kernel<<<block1D_j, threads1D>>>(i_max, j_max, unified_u, unified_v, LEFT);
        set_noslip_linear_kernel<<<block1D_j, threads1D>>>(i_max, j_max, unified_u, unified_v, RIGHT);
        set_noslip_linear_kernel<<<block1D_i, threads1D>>>(i_max, j_max, unified_u, unified_v, BOTTOM);
        set_inflow_linear_kernel<<<block1D_i, threads1D>>>(i_max, j_max, unified_u, unified_v, TOP, 1.0, 0.0);
    }
    else if (problem == 2){
        set_noslip_linear_kernel<<<block1D_j, threads1D>>>(i_max, j_max, unified_u, unified_v, LEFT);
        set_noslip_linear_kernel<<<block1D_j, threads1D>>>(i_max, j_max, unified_u, unified_v, RIGHT);
        set_noslip_linear_kernel<<<block1D_i, threads1D>>>(i_max, j_max, unified_u, unified_v, BOTTOM);
        set_inflow_linear_kernel<<<block1D_i, threads1D>>>(i_max, j_max, unified_u, unified_v, TOP, sin(f*(*t)), 0.0);           
    }
    else {
        return -1;
    }
    CUDACHECK(hipDeviceSynchronize());
    printf("Conditions set!\n");

    // 2. FG calculation (GPU)
    dim3 block2D(16, 16);
    dim3 grid2D((i_max+block2D.x-1)/block2D.x, (j_max+block2D.y-1)/block2D.y);
    FG_linear_kernel<<<grid2D, block2D>>>(unified_u, unified_v, unified_F, unified_G, i_max, j_max, Re, g_x, g_y, delta_t, delta_x, delta_y, gamma_factor);
    CUDACHECK(hipDeviceSynchronize());
    printf("F, G calculated!\n");

    // 3. RHS calculation (GPU)
    RHS_kernel<<<grid2D, block2D>>>(unified_F, unified_G, unified_RHS, i_max, j_max, delta_t, delta_x, delta_y);
    CUDACHECK(hipDeviceSynchronize());
    printf("RHS calculated!\n");

    // 4. Copy p, RHS, res to unified arrays (GPU, if needed)
    // (If all arrays are already on GPU, this step can be skipped)

    // 5. SOR loop (already uses GPU kernels)
    dim3 blockSOR(16, 16);
    dim3 gridSOR((i_max+blockSOR.x-1)/blockSOR.x, (j_max+blockSOR.y-1)/blockSOR.y);
    while (it < max_it) {
        // Atualizar condições de contorno de pressão (ghost cells) na GPU
        // Kernel para copiar bordas (pode ser otimizado, mas aqui faz em CPU para simplicidade)
        for (int i = 1; i <= i_max; i++) {
            unified_p[i * (j_max + 2) + 0] = unified_p[i * (j_max + 2) + 1];
            unified_p[i * (j_max + 2) + (j_max + 1)] = unified_p[i * (j_max + 2) + j_max];
        }
        for (int j = 1; j <= j_max; j++) {
            unified_p[0 * (j_max + 2) + j] = unified_p[1 * (j_max + 2) + j];
            unified_p[(i_max + 1) * (j_max + 2) + j] = unified_p[i_max * (j_max + 2) + j];
        }
        CUDACHECK(hipDeviceSynchronize());
        // Red points
        RedSORKernel<<<gridSOR, blockSOR>>>(unified_p, unified_RHS, i_max, j_max, omega, dxdx, dydy);
        CUDACHECK(hipGetLastError());
        CUDACHECK(hipDeviceSynchronize());
        // Black points
        BlackSORKernel<<<gridSOR, blockSOR>>>(unified_p, unified_RHS, i_max, j_max, omega, dxdx, dydy);
        CUDACHECK(hipGetLastError());
        CUDACHECK(hipDeviceSynchronize());
        // Calcular resíduos
        CalculateResidualKernel<<<gridSOR, blockSOR>>>(unified_p, unified_res, unified_RHS, i_max, j_max, dxdx, dydy);
        CUDACHECK(hipGetLastError());
        CUDACHECK(hipDeviceSynchronize());
        // Verificar convergência (CPU reduction)
        double res_norm = 0.0;
        for (int i = 1; i <= i_max; i++) {
            for (int j = 1; j <= j_max; j++) {
                res_norm += unified_res[i * (j_max + 2) + j] * unified_res[i * (j_max + 2) + j];
            }
        }
        res_norm = sqrt(res_norm / (i_max * j_max));
        if (res_norm <= eps * (norm_p + 0.01)) {
            break; // Convergência atingida
        }
        it++;
    }
    // Copiar resultados de volta para os arrays 2D originais
    for (int i = 0; i <= i_max + 1; i++) {
        for (int j = 0; j <= j_max + 1; j++) {
            p[i][j] = unified_p[i * (j_max + 2) + j];
        }
    }
    printf("SOR complete!\n");
    // 4. Atualizar u e v usando kernel update_uv_kernel
    update_uv_kernel<<<grid2D, block2D>>>(unified_u, unified_v, unified_F, unified_G, unified_p, i_max, j_max, delta_t, delta_x, delta_y);
    KERNEL_CHECK("update_uv_kernel");
    CUDACHECK(hipDeviceSynchronize());
    
    // Não há mais hipMalloc/hipFree aqui!
    // Corrigir acesso incorreto para os valores centrais
    int center_i = i_max/2;
    int center_j = j_max/2;
    
    // Verificar limites antes de acessar valores centrais
    if (center_i < 0 || center_i > i_max+1 || center_j < 0 || center_j > j_max+1) {
        return -1;
    }
    
    int center_idx = center_i * (j_max + 2) + center_j;
    
    // Output central values for debugging
    printf("TIMESTEP: %d TIME: %.6f\n", (*n_out), *t);
    printf("U-CENTER: %.6f\n", unified_u[center_idx]);
    printf("V-CENTER: %.6f\n", unified_v[center_idx]);
    printf("P-CENTER: %.6f\n", unified_p[center_idx]);
    
    (*n_out)++;  // Incrementa o valor apontado pelo ponteiro
    *t += delta_t;  // Atualiza o tempo

    return (it < max_it) ? 0 : -1;
}

// IMPORTANTE: Versão template removida para evitar conflito com __double_as_int
// Usamos apenas a versão especializada para double abaixo

// Kernel para encontrar o valor máximo absoluto em uma matriz linearizada (versão especializada para double)
__global__ void max_mat_kernel_double(const double* mat, int i_max, int j_max, double* max_val) {
    // Verificações de parâmetros
    if (mat == NULL || max_val == NULL || i_max <= 0 || j_max <= 0) {
        // Na GPU não podemos fazer muito mais do que isso
        return;
    }
    
    __shared__ double sdata[256];
    int tid = threadIdx.x;
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    int total = (i_max) * (j_max);
    double local_max = 0.0;
    
    // Encontrar máximo local
    for (int k = idx; k < total; k += blockDim.x * gridDim.x) {
        int i = 1 + k / j_max;
        int j = 1 + k % j_max;
        
        // Verificação de limites
        if (i >= 0 && i <= i_max+1 && j >= 0 && j <= j_max+1) {
            double val = fabs(mat[i * (j_max + 2) + j]);
            if (val > local_max) local_max = val;
        }
    }
    
    // Armazenar em memória compartilhada
    sdata[tid] = local_max;
    __syncthreads();
    
    // Redução em bloco
    for (int s = blockDim.x / 2; s > 0; s >>= 1) {
        if (tid < s) {
            if (sdata[tid + s] > sdata[tid]) sdata[tid] = sdata[tid + s];
        }
        __syncthreads();
    }
    
    // Apenas a thread 0 do bloco atualiza o máximo global
    if (tid == 0) {
        double old_val = *max_val;
        double my_val = sdata[0];
        
        // Usar atomicCAS para atualização segura do máximo
        while (my_val > old_val) {
            double assumed = old_val;
            
            // Usar union para conversão de tipo segura
            union { double d; unsigned long long int i; } old_union, new_union;
            old_union.d = old_val;
            new_union.d = my_val;
            
            // Operação atômica para atualizar o máximo global
            old_val = atomicCAS((unsigned long long int*)max_val, 
                                old_union.i, 
                                new_union.i);
                                
            // Se o valor não mudou desde nossa última leitura, podemos sair
            if (old_val == assumed) {
                break;
            }
        }
    }
}

// Kernel para calcular dp_dx e dp_dy em arrays de saída
__global__ void dp_dx_kernel(const double* p, double* out, int i_max, int j_max, double delta_x) {
    int i = blockIdx.x * blockDim.x + threadIdx.x + 1;
    int j = blockIdx.y * blockDim.y + threadIdx.y + 1;
    if (i <= i_max && j <= j_max) {
        out[i * (j_max + 2) + j] = (p[i * (j_max + 2) + j+1] - p[i * (j_max + 2) + j]) / delta_x;
    }
}
__global__ void dp_dy_kernel(const double* p, double* out, int i_max, int j_max, double delta_y) {
    int i = blockIdx.x * blockDim.x + threadIdx.x + 1;
    int j = blockIdx.y * blockDim.y + threadIdx.y + 1;
    if (i <= i_max && j <= j_max) {
        out[i * (j_max + 2) + j] = (p[(i+1) * (j_max + 2) + j] - p[i * (j_max + 2) + j]) / delta_y;
    }
}

// Kernel para set_noslip_linear
__global__ void set_noslip_linear_kernel(int i_max, int j_max, double* u, double* v, int side) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x + 1;
    if (side == LEFT && idx <= j_max) {
        u[0 * (j_max + 2) + idx] = 0.0;
        v[0 * (j_max + 2) + idx] = -v[1 * (j_max + 2) + idx];
    } else if (side == RIGHT && idx <= j_max) {
        u[i_max * (j_max + 2) + idx] = 0.0;
        v[(i_max+1) * (j_max + 2) + idx] = -v[i_max * (j_max + 2) + idx];
    } else if (side == TOP && idx <= i_max) {
        u[idx * (j_max + 2) + j_max+1] = -u[idx * (j_max + 2) + j_max];
        v[idx * (j_max + 2) + j_max] = 0.0;
    } else if (side == BOTTOM && idx <= i_max) {
        u[idx * (j_max + 2) + 0] = -u[idx * (j_max + 2) + 1];
        v[idx * (j_max + 2) + 0] = 0.0;
    }
}

// Kernel para set_inflow_linear
__global__ void set_inflow_linear_kernel(int i_max, int j_max, double* u, double* v, int side, double u_fix, double v_fix) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x + 1;
    if (side == TOP && idx <= i_max) {
        u[idx * (j_max + 2) + j_max+1] = 2 * u_fix - u[idx * (j_max + 2) + j_max];
        v[idx * (j_max + 2) + j_max] = v_fix;
    } else if (side == BOTTOM && idx <= i_max) {
        u[idx * (j_max + 2) + 0] = 2 * u_fix - u[idx * (j_max + 2) + 1];
        v[idx * (j_max + 2) + 0] = v_fix;
    } else if (side == LEFT && idx <= j_max) {
        u[0 * (j_max + 2) + idx] = u_fix;
        v[0 * (j_max + 2) + idx] = 2 * v_fix - v[1 * (j_max + 2) + idx];
    } else if (side == RIGHT && idx <= j_max) {
        u[i_max * (j_max + 2) + idx] = u_fix;
        v[(i_max+1) * (j_max + 2) + idx] = 2 * v_fix - v[i_max * (j_max + 2) + idx];
    }
}

// Kernel para calcular F e G (Navier-Stokes)
__global__ void FG_linear_kernel(double* u, double* v, double* F, double* G, int i_max, int j_max, double Re, double g_x, double g_y, double delta_t, double delta_x, double delta_y, double gamma_factor) {
    int i = blockIdx.x * blockDim.x + threadIdx.x + 1;
    int j = blockIdx.y * blockDim.y + threadIdx.y + 1;
    if (i <= i_max - 1 && j <= j_max) {
        // F (u preditor)
        double du2dx = ((u[(i+1)*(j_max+2)+j] + u[i*(j_max+2)+j])*(u[(i+1)*(j_max+2)+j] + u[i*(j_max+2)+j])
                        - (u[i*(j_max+2)+j] + u[(i-1)*(j_max+2)+j])*(u[i*(j_max+2)+j] + u[(i-1)*(j_max+2)+j])) / (4.0*delta_x);
        double duvdy = ((v[i*(j_max+2)+j] + v[(i+1)*(j_max+2)+j])*(u[i*(j_max+2)+j+1] + u[i*(j_max+2)+j])
                        - (v[i*(j_max+2)+j-1] + v[(i+1)*(j_max+2)+j-1])*(u[i*(j_max+2)+j] + u[i*(j_max+2)+j-1])) / (4.0*delta_y);
        double laplu = (u[(i+1)*(j_max+2)+j] - 2.0*u[i*(j_max+2)+j] + u[(i-1)*(j_max+2)+j]) / (delta_x*delta_x)
                        + (u[i*(j_max+2)+j+1] - 2.0*u[i*(j_max+2)+j] + u[i*(j_max+2)+j-1]) / (delta_y*delta_y);
        F[i*(j_max+2)+j] = u[i*(j_max+2)+j] + delta_t * ((laplu/Re) - du2dx - duvdy + g_x);
    }
    if (i <= i_max && j <= j_max - 1) {
        // G (v preditor)
        double dv2dy = ((v[i*(j_max+2)+j+1] + v[i*(j_max+2)+j])*(v[i*(j_max+2)+j+1] + v[i*(j_max+2)+j])
                        - (v[i*(j_max+2)+j] + v[i*(j_max+2)+j-1])*(v[i*(j_max+2)+j] + v[i*(j_max+2)+j-1])) / (4.0*delta_y);
        double duvdx = ((u[i*(j_max+2)+j] + u[i*(j_max+2)+j+1])*(v[(i+1)*(j_max+2)+j] + v[i*(j_max+2)+j])
                        - (u[(i-1)*(j_max+2)+j] + u[(i-1)*(j_max+2)+j+1])*(v[i*(j_max+2)+j] + v[(i-1)*(j_max+2)+j])) / (4.0*delta_x);
        double laplv = (v[(i+1)*(j_max+2)+j] - 2.0*v[i*(j_max+2)+j] + v[(i-1)*(j_max+2)+j]) / (delta_x*delta_x)
                        + (v[i*(j_max+2)+j+1] - 2.0*v[i*(j_max+2)+j] + v[i*(j_max+2)+j-1]) / (delta_y*delta_y);
        G[i*(j_max+2)+j] = v[i*(j_max+2)+j] + delta_t * ((laplv/Re) - dv2dy - duvdx + g_y);
    }
}

// Kernel para calcular RHS
__global__ void RHS_kernel(double* F, double* G, double* RHS, int i_max, int j_max, double delta_t, double delta_x, double delta_y) {
    int i = blockIdx.x * blockDim.x + threadIdx.x + 1;
    int j = blockIdx.y * blockDim.y + threadIdx.y + 1;
    if (i <= i_max && j <= j_max) {
        RHS[i*(j_max+2)+j] = ((F[i*(j_max+2)+j] - F[(i-1)*(j_max+2)+j]) / delta_x
                            + (G[i*(j_max+2)+j] - G[i*(j_max+2)+j-1]) / delta_y) / delta_t;
    }
}

// Kernel para atualizar u e v após SOR
__global__ void update_uv_kernel(double* u, double* v, double* F, double* G, double* p, int i_max, int j_max, double delta_t, double delta_x, double delta_y) {
    int i = blockIdx.x * blockDim.x + threadIdx.x + 1;
    int j = blockIdx.y * blockDim.y + threadIdx.y + 1;
    if (i <= i_max - 1 && j <= j_max) {
        u[i*(j_max+2)+j] = F[i*(j_max+2)+j] - delta_t * (p[(i+1)*(j_max+2)+j] - p[i*(j_max+2)+j]) / delta_x;
    }
    if (i <= i_max && j <= j_max - 1) {
        v[i*(j_max+2)+j] = G[i*(j_max+2)+j] - delta_t * (p[i*(j_max+2)+j+1] - p[i*(j_max+2)+j]) / delta_y;
    }
}

// Função utilitária para mínimo de até 4 valores double
__host__ __device__ double n_min(int n, double a, double b, double c, double d) {
    double minval = a;
    if (b < minval) minval = b;
    if (c < minval) minval = c;
    if (d < minval) minval = d;
    return minval;
}

// Exemplo de chamada de kernel para max_mat:
// double* d_max;
// hipMallocManaged(&d_max, sizeof(double));
// *d_max = 0.0;
// max_mat_kernel<<<numBlocks, blockSize>>>(unified_u, i_max, j_max, d_max);
// hipDeviceSynchronize();
// double max_val = *d_max;
// hipFree(d_max);
//
// Para os outros kernels, use grid/block adequados conforme o tamanho do domínio.
