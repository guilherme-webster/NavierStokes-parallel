#include "hip/hip_runtime.h"
#include "kernel.h"
#include "vector"
#include <stdlib.h>
#include <stdio.h>
#include <hip/hip_runtime.h>

typedef struct {
    int i, j;
    int side; // TOP=0, BOTTOM=1, LEFT=2, RIGHT=3
} BoundaryPoint;

double du_max,dv_max;
double* d_u, d_v, d_p;
double d_delta_t, d_delta_x, d_delta_y;
double d_tau, d_gamma,d_Re;
int* d_boundary_index;
BoundaryPoint* d_boundary_indices;
int di_max, dj_max;
double* d_F, d_G;
double* d_RHS;
double dg_x, dg_y;

void init_memory(int i_max, int j_max, double* delta_t, double delta_x, double delta_y, double Re, BoundaryPoint* h_boundary_indices) {
    size_t size = (i_max + 2) * (j_max + 2) * sizeof(double);
    hipMalloc((void**)&du_max * sizeof(double));
    hipMalloc((void**)&dv_max * sizeof(double));
    hipMalloc((void**) d_u * size);
    hipMalloc((void**) d_v * size);
    hipMalloc((void**) d_p * size);
    hipMalloc((void*) d_Re * sizeof(double));
    hipMalloc((void*) d_tau * sizeof(double));
    hipMalloc((void*) d_gamma * sizeof(double));
    hipMalloc((void*) d_delta_t * sizeof(double));
    hipMalloc((void*) d_delta_x * sizeof(double));
    hipMalloc((void*) d_delta_y * sizeof(double));
    hipMalloc((void*) d_boundary_index * sizeof(int));
    hipMalloc((void**)&d_boundary_indices, total_points * sizeof(BoundaryPoint));
    hipMalloc((void*)Re * sizeof(double));
    hipMalloc((void**)d_RHS * size);
    hipMalloc((void**)d_F * size);
    hipMalloc((void**)d_G * size);
    hipMalloc((void*) d_gy * sizeof(double));
    hipMalloc((void*) d_gx * sizeof(double));

    hipMemcpy(d_boundary_indices, h_boundary_indices, total_points * sizeof(BoundaryPoint), hipMemcpyHostToDevice);
    hipMemset(d_u,0, sizeof(double));
    hipMemset(d_v,0, sizeof(double));
    hipMemset(d_p,0, sizeof(double));
    hipMemset(dv_max,0, sizeof(double));
    hipMemset(du_max,0, sizeof(double));
    hipMemset(d_Re,0, sizeof(double));
    hipMemset(d_tau,0, sizeof(double));
    hipMemset(d_gamma,0, sizeof(double));
    hipMemset(d_delta_t,0, sizeof(double));
    hipMemset(d_delta_x,0, sizeof(double));
    hipMemset(d_delta_y,0, sizeof(double));
    hipMemset(d_boundary_index,0, sizeof(int));
    hipMemset(d_boundary_indices,0, sizeof(BoundaryPoint));
    hipMemset(d_F,0, sizeof(double));
    hipMemset(d_G,0, sizeof(double));
    hipMemset(d_RHS,0, sizeof(double));
    hipMemset(dg_x,0, sizeof(double));
    hipMemset(dg_y,0, sizeof(double));
}



void orquestration(double** u, double** v, double** p, double** res, double** RHS, double** F, double** G,
    int i_max, int j_max, double* delta_t, double delta_x, double delta_y, double Re, double g_x, double g_y,
    double tau, double omega, double epsilon, int max_it, int n_print) {
    
    int threads = 256;
    int blocks = (i_max * j_max + threads - 1) / threads;
    int size = i_max * j_max;
    extern __shared__ double shared_data[];
    di_max = i_max;
    dj_max = j_max;
    // acha o máximo da matriz u e v
    while (size > 1){
        blocks = (size + threads - 1) / threads;

        max_reduce_kernel<<<blocks, threads, threads * sizeof(double)>>>(i_max, j_max, u, d_u);
        max_reduce_kernel<<<blocks, threads, threads * sizeof(double)>>>(i_max, j_max, v, d_v);
        hipDeviceSynchronize();
    }
    
    min_and_gamma<<<1, 1>>>();
    
    hipDeviceSynchronize();

    update_boundaries_kernel<<<blocks, threads>>>();

    hipDeviceSynchronize();

    printf("Conditions set!\n");

    // now we calculate F and G
    calculate_F<<<blocks, threads>>>(d_F, d_u, d_v, di_max, dj_max, d_Re, dg_x, d_delta_t, d_delta_x, d_delta_y, d_gamma);
    calculate_G<<<blocks, threads>>>(d_G, d_u, d_v, di_max, dj_max, d_Re, dg_y, d_delta_t, d_delta_x, d_delta_y, d_gamma);    

    hipDeviceSynchronize();

    printf("F, G calculated!\n");

    // now we calculate rhs
    calculate_RHS<<<blocks, threads>>>(RHS, F, G, d_u, d_v, di_max, dj_max, d_delta_t, d_delta_x, d_delta_y);

    hipDeviceSynchronize();
    
    L2_norm<<<blocks, threads>>>(d_norm_p, d_p, di_max, dj_max);
    
    hipDeviceSynchronize();
    
    hipMemcpy(&norm_p, d_norm_p, sizeof(double), hipMemcpyDeviceToHost);
    double norm = sqrt(norm_p/ ((i_max) * (j_max)));

    while(n < max_it) {
        calculate_ghost<<<blocks, threads>>>();

        hipDeviceSynchronize();

        printf("RHS calculated!\n");
        // Now execute de SOR black and red
        red_kernel<<<blocks, threads>>>(d_p, d_RHS, d_u, d_v, di_max, dj_max, d_delta_x, d_delta_y, omega);
        
        hipDeviceSynchronize();

        black_kernel<<<blocks, threads>>>(d_p, d_RHS, d_u, d_v, di_max, dj_max, d_delta_x, d_delta_y, omega);

        hipDeviceSynchronize();
    
        residual_kernel<<<blocks, threads>>>(d_res, d_p, d_RHS, di_max, dj_max, d_delta_x, d_delta_y);
        
        hipDeviceSynchronize();

        L2_norm<<<blocks, threads>>>(d_norm_res, d_res, di_max, dj_max);
        hipDeviceSynchronize();
        hipMemcpy(&norm_res, d_norm_res, sizeof(double), hipMemcpyDeviceToHost);
        double temp = sqrt(norm_res / ((i_max) * (j_max)));
        if(temp <= epsilon * (norm + 0.01)) {
            return 0;
        }
    }

    printf("SOR complete!\n");
    update_velocity_kernel<<<blocks, threads>>>(d_u, d_v, d_p, di_max, dj_max, d_delta_t, d_delta_x, d_delta_y);
    hipDeviceSynchronize();
    printf("Velocities updated!\n");
    // update the velocities

    double result[3];
    extract_value_kernel<<<1, 1>>>(d_u, d_v, d_p, i_max, j_max, result);
    hipDeviceSynchronize();

    printf("U-CENTER: %.6f\n", result[0]);
    printf("V-CENTER: %.6f\n", result[1]);
    printf("P-CENTER: %.6f\n", result[2]);
    
}


__global__ void min_and_gamma (){
    double min = fmin(Re / 2.0 / ( 1.0 / d_delta_x / d_delta_x + 1.0 / d_delta_y / d_delta_y ), d_delta_x / fabs(du_max));
    min = fmin(min, d_delta_y / fabs(dv_max));
    min = fmin(min, 3.0);
    d_delta_t = tau * min;
    d_gamma = fmax(du_max * d_delta_t / d_delta_x, dv_max * d_delta_t / d_delta_y);
}


__global__ void max_reduce_kernel(int i_max, int j_max, double* arr, double* max_val) {
    extern __shared__ double shared_data[];
    int tid = threadIdx.x;
    int global_idx = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;
    double max_val_local = 0.0;

    for (int i = global_idx; i < i_max * j_max; i += stride) {
        if (arr[i] > max_val_local) {
            max_val_local = arr[i];
        }
    }

    shared_data[tid] = max_val_local;
    __syncthreads();

    for (int s = blockDim.x / 2; s > 0; s >>= 1) {
        if (tid < s && shared_data[tid + s] > shared_data[tid]) {
            shared_data[tid] = shared_data[tid + s];
        }
        __syncthreads();
    }

    if (tid == 0) {
        atomicMax(max_val, shared_data[0]);
    }
}


BoundaryPoint* generate_boundary_indices(int i_max, int j_max, int* total_points) {
    *total_points = 2 * (i_max + j_max);
    BoundaryPoint* h_boundary_indices = (BoundaryPoint*)malloc(*total_points * sizeof(BoundaryPoint));
    int idx = 0;

    // Borda TOP (j = j_max)
    for (int i = 1; i <= i_max; i++) {
        h_boundary_indices[idx++] = (BoundaryPoint){i, j_max, 0};
    }

    // Borda BOTTOM (j = 0)
    for (int i = 1; i <= i_max; i++) {
        h_boundary_indices[idx++] = (BoundaryPoint){i, 0, 1};
    }

    // Borda LEFT (i = 0)
    for (int j = 1; j <= j_max; j++) {
        h_boundary_indices[idx++] = (BoundaryPoint){0, j, 2};
    }

    // Borda RIGHT (i = i_max + 1)
    for (int j = 1; j <= j_max; j++) {
        h_boundary_indices[idx++] = (BoundaryPoint){i_max + 1, j, 3};
    }

    return h_boundary_indices;
}

__global__ void update_boundaries_kernel() {
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    if (tid >= 2 * (di_max + dj_max)) return;

    BoundaryPoint point = d_boundary_indices[tid];
    int i = point.i;
    int j = point.j;
    // O vfix e ufix são fixos pois tratam apenas do caso 1 do simulador
    switch (point.side) {
        case 0: // TOP
            d_v[i * (dj_max + 1) + j] = 0.0;
            d_u[i * (dj_max + 2) + (j + 1)] = 2 * 1.0 - d_u[i * (dj_max + 2) + j];
            break;
        case 1: // BOTTOM
            d_v[i * (dj_max + 1) + j] = 0.0;
            d_u[i * (dj_max + 2) + j] = 2 * 0.0 - d_u[i * (dj_max + 2) + (j + 1)];
            break;
        case 2: // LEFT
            d_u[i * (dj_max + 2) + j] = 0.0;
            d_v[i * (dj_max + 1) + j] = 2 * 0.0 - d_v[(i + 1) * (dj_max + 1) + j];
            break;
        case 3: // RIGHT
            d_u[i * (dj_max + 2) + j] = 0.0;
            d_v[i * (dj_max + 1) + j] = 2 * 0.0 - d_v[(i - 1) * (dj_max + 1) + j];
            break;
    }
}

// Funções diferenças finitas com índices linearizados para GPU

__device__ double du2_dx(double* u, double* v, int i, int j, double delta_x, double gamma, int j_max) {
    // Índices lineares
    int idx = i * (j_max + 2) + j;
    int idx_i_plus = (i+1) * (j_max + 2) + j;
    int idx_i_minus = (i-1) * (j_max + 2) + j;
    
    double stencil1 = 0.5 * (u[idx] + u[idx_i_plus]);
    double stencil2 = 0.5 * (u[idx_i_minus] + u[idx]);

    double stencil3 = fabs(stencil1) * 0.5 * (u[idx] - u[idx_i_plus]);
    double stencil4 = fabs(stencil2) * 0.5 * (u[idx_i_minus] - u[idx]);

    return 1/delta_x * (stencil1*stencil1 - stencil2*stencil2) + gamma / delta_x * (stencil3 - stencil4);
}

__device__ double duv_dy(double* u, double* v, int i, int j, double delta_y, double gamma, int j_max) {
    // Índices lineares para u (com j_max + 2 colunas)
    int u_idx = i * (j_max + 2) + j;
    int u_idx_j_plus = i * (j_max + 2) + (j+1);
    int u_idx_j_minus = i * (j_max + 2) + (j-1);
    
    // Índices lineares para v (com j_max + 1 colunas) 
    int v_idx = i * (j_max + 1) + j;
    int v_idx_i_plus = (i+1) * (j_max + 1) + j;
    int v_idx_j_minus = i * (j_max + 1) + (j-1);
    int v_idx_i_plus_j_minus = (i+1) * (j_max + 1) + (j-1);

    double stencil1 = 0.5 * (v[v_idx] + v[v_idx_i_plus]);
    double stencil2 = 0.5 * (v[v_idx_j_minus] + v[v_idx_i_plus_j_minus]);

    double stencil3 = stencil1 * 0.5 * (u[u_idx] + u[u_idx_j_plus]);
    double stencil4 = stencil2 * 0.5 * (u[u_idx_j_minus] + u[u_idx]);

    double stencil5 = fabs(stencil1) * 0.5 * (u[u_idx] - u[u_idx_j_plus]);
    double stencil6 = fabs(stencil2) * 0.5 * (u[u_idx_j_minus] - u[u_idx]);

    return 1/delta_y * (stencil3 - stencil4) + gamma / delta_y * (stencil5 - stencil6);
}

__device__ double dv2_dy(double* v, double* u, int i, int j, double delta_y, double gamma, int j_max) {
    // Índices lineares para v
    int idx = i * (j_max + 1) + j;
    int idx_j_plus = i * (j_max + 1) + (j+1);
    int idx_j_minus = i * (j_max + 1) + (j-1);
    
    double stencil1 = 0.5 * (v[idx] + v[idx_j_plus]);
    double stencil2 = 0.5 * (v[idx_j_minus] + v[idx]);

    double stencil3 = fabs(stencil1) * 0.5 * (v[idx] - v[idx_j_plus]);
    double stencil4 = fabs(stencil2) * 0.5 * (v[idx_j_minus] - v[idx]);

    return 1/delta_y * (stencil1*stencil1 - stencil2*stencil2) + gamma / delta_y * (stencil3 - stencil4);
}

__device__ double duv_dx(double* u, double* v, int i, int j, double delta_x, double gamma, int j_max) {
    // Índices lineares para u
    int u_idx = i * (j_max + 2) + j;
    int u_idx_j_plus = i * (j_max + 2) + (j+1);
    int u_idx_i_minus = (i-1) * (j_max + 2) + j;
    int u_idx_i_minus_j_plus = (i-1) * (j_max + 2) + (j+1);
    
    // Índices lineares para v
    int v_idx = i * (j_max + 1) + j;
    int v_idx_i_plus = (i+1) * (j_max + 1) + j;
    int v_idx_i_minus = (i-1) * (j_max + 1) + j;

    double stencil1 = 0.5 * (u[u_idx] + u[u_idx_j_plus]);
    double stencil2 = 0.5 * (u[u_idx_i_minus] + u[u_idx_i_minus_j_plus]);

    double stencil3 = stencil1 * 0.5 * (v[v_idx] + v[v_idx_i_plus]);
    double stencil4 = stencil2 * 0.5 * (v[v_idx_i_minus] + v[v_idx]);

    double stencil5 = fabs(stencil1) * 0.5 * (v[v_idx] - v[v_idx_i_plus]);
    double stencil6 = fabs(stencil2) * 0.5 * (v[v_idx_i_minus] - v[v_idx]);

    return 1/delta_x * (stencil3 - stencil4) + gamma / delta_x * (stencil5 - stencil6);
}

/**
 * Central differences for second derivatives.
 */

__device__ double d2u_dx2(double* u, int i, int j, double delta_x, int j_max) {
    int idx = i * (j_max + 2) + j;
    int idx_i_plus = (i+1) * (j_max + 2) + j;
    int idx_i_minus = (i-1) * (j_max + 2) + j;
    
    return (u[idx_i_plus] - 2 * u[idx] + u[idx_i_minus]) / (delta_x*delta_x);
}

__device__ double d2u_dy2(double* u, int i, int j, double delta_y, int j_max) {
    int idx = i * (j_max + 2) + j;
    int idx_j_plus = i * (j_max + 2) + (j+1);
    int idx_j_minus = i * (j_max + 2) + (j-1);
    
    return (u[idx_j_plus] - 2 * u[idx] + u[idx_j_minus]) / (delta_y*delta_y);
}

__device__ double d2v_dx2(double* v, int i, int j, double delta_x, int j_max) {
    int idx = i * (j_max + 1) + j;
    int idx_i_plus = (i+1) * (j_max + 1) + j;
    int idx_i_minus = (i-1) * (j_max + 1) + j;
    
    return (v[idx_i_plus] - 2 * v[idx] + v[idx_i_minus]) / (delta_x*delta_x);
}

__device__ double d2v_dy2(double* v, int i, int j, double delta_y, int j_max) {
    int idx = i * (j_max + 1) + j;
    int idx_j_plus = i * (j_max + 1) + (j+1);
    int idx_j_minus = i * (j_max + 1) + (j-1);
    
    return (v[idx_j_plus] - 2 * v[idx] + v[idx_j_minus]) / (delta_y*delta_y);
}

__global__ void calculate_F(double* F, double* u, double* v, int i_max, int j_max, double Re,
    double g_x, double g_y, double delta_t, double delta_x, double delta_y, double gamma) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int j = blockIdx.y * blockDim.y + threadIdx.y;
    // u and v must be d_u and d_v when this function is called
    if (i > 0 && i <= di_max && j > 0 && j <= dj_max) {
        F[i * (dj_max + 2) + j] = u[i * (dj_max + 2) + j] + delta_t * ((1/Re) * (d2u_dx2(u, i, j, delta_x) + d2u_dy2(u, i, j, delta_y)) - du2_dx(u, v, i, j, delta_x, gamma) - duv_dy(u, v, i, j, delta_y, gamma) + g_x);
    }
}

__global__ void calculate_G(double * G, double* u, double* v, int i_max, int j_max, double Re,
    double g_x, double g_y, double delta_t, double delta_x, double delta_y, double gamma) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int j = blockIdx.y * blockDim.y + threadIdx.y;
    // u and v must be d_u and d_v when this function is called
    if (i > 0 && i <= di_max && j > 0 && j <= dj_max) {
        // +1 ou +2
        G[i * (dj_max + 1) + j] = v[i * (dj_max + 1) + j] + delta_t * ((1/Re) * (d2v_dx2(v, i, j, delta_x) + d2v_dy2(v, i, j, delta_y)) - duv_dx(u, v, i, j, delta_x, gamma) - dv2_dy(v, u, i, j, delta_y, gamma) + g_y);
    }
}


__global__ void calculate_RHS(double* RHS, double* F, double* G, double* u, double* v, int i_max, int j_max,
    double delta_t, double delta_x, double delta_y) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int j = blockIdx.y * blockDim.y + threadIdx.y;

    if (i > 0 && i <= di_max && j > 0 && j <= dj_max) {
        RHS[i * (dj_max + 2) + j] = 1.0 / delta_t * ((F[i * (dj_max + 2) + j] - F[(i-1) * (dj_max + 2) + j]) / delta_x + (G[i * (dj_max + 1) + j] - G[i * (dj_max + 1) + (j-1)]) / delta_y);
    }
}

__global__ void red_kernel(double* p, double* RHS, double* u, double* v, int i_max, int j_max,
    double delta_x, double delta_y, double omega) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int j = blockIdx.y * blockDim.y + threadIdx.y;
    double dxdx = delta_x * delta_x;
    double dydy = delta_y * delta_y;
    if (i > 0 && i <= di_max && j > 0 && j <= dj_max) {
        if ((i + j) % 2 == 0) {
            p[i * (dj_max + 2) + j] = (1 - omega) * p[i * (dj_max + 2) + j] +
                omega / (2.0 * (1.0/ dxdx + 1.0 /dydy))*
                ((p[(i+1) * (dj_max + 2) + j] + p[(i-1) * (dj_max + 2) + j]) / dxdx + (p[i * (dj_max + 2) + (j+1)] + p[i * (dj_max + 2) + (j-1)]) / dydy -
                RHS[i * (dj_max + 2) + j]);
        }
    }
}


__global__ void black_kernel(double* p, double* RHS, double* u, double* v, int i_max, int j_max,
    double delta_x, double delta_y, double omega) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int j = blockIdx.y * blockDim.y + threadIdx.y;

    if (i > 0 && i <= di_max && j > 0 && j <= dj_max) {
        if ((i + j) % 2 == 1) {
            p[i * (dj_max + 2) + j] = (1 - omega) * p[i * (dj_max + 2) + j] +
                omega / (2.0 * (1.0/ dxdx + 1.0 /dydy))*
                ((p[(i+1) * (dj_max + 2) + j] + p[(i-1) * (dj_max + 2) + j]) / dxdx + (p[i * (dj_max + 2) + (j+1)] + p[i * (dj_max + 2) + (j-1)]) / dydy -
                RHS[i * (dj_max + 2) + j]);
        }
    }
}


__global__ void calculate_ghost() {
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    if (tid >= 2 * (di_max + dj_max)) return;
    
    BoundaryPoint point = d_boundary_indices[tid];
    int i = point.i;
    int j = point.j;
    
    // Tratar condições de contorno de Neumann para pressão (gradiente zero)
    switch (point.side) {
        case 0: // TOP (j = j_max)
            // p[i][j_max+1] = p[i][j_max]
            d_p[i * (dj_max + 2) + (j+1)] = d_p[i * (dj_max + 2) + j];
            break;
            
        case 1: // BOTTOM (j = 0)
            // p[i][0] = p[i][1]
            d_p[i * (dj_max + 2) + 0] = d_p[i * (dj_max + 2) + 1];
            break;
            
        case 2: // LEFT (i = 0)
            // p[0][j] = p[1][j]
            d_p[0 * (dj_max + 2) + j] = d_p[1 * (dj_max + 2) + j];
            break;
            
        case 3: // RIGHT (i = i_max+1)
            // p[i_max+1][j] = p[i_max][j]
            d_p[(i_max + 1) * (dj_max + 2) + j] = d_p[i_max * (dj_max + 2) + j];
            break;
    }
}


__global__ void L2_norm(double* norm, double* m, int i_max, int j_max) {
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    if (tid >= i_max * j_max) return;

    double value = m[tid];
    atomicAdd(norm, value * value);
}


__global__ void residual_kernel(double* res, double* p, double* RHS, int i_max, int j_max,
    double delta_x, double delta_y) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int j = blockIdx.y * blockDim.y + threadIdx.y;

    if (i > 0 && i <= di_max && j > 0 && j <= dj_max) {
        res[i * (dj_max + 2) + j] = ((p[(i+1) * (dj_max + 2) + j] - 2 * p[i * (dj_max + 2) + j] + p[(i-1) * (dj_max + 2) + j]) / (delta_x * delta_x) +
            (p[i * (dj_max + 2) + (j+1)] - 2 * p[i * (dj_max + 2) + j] + p[i * (dj_max + 2) + (j-1)]) / (delta_y * delta_y)) - RHS[i * (dj_max + 2) + j];
    }
}

__global__ void update_velocity_kernel(double* u, double* v, double* p, int i_max, int j_max,
    double delta_t, double delta_x, double delta_y) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int j = blockIdx.y * blockDim.y + threadIdx.y;

    if (i > 0 && i <= di_max && j > 0 && j <= dj_max) {
        if (i <= di_max - 1) u[i * (dj_max + 2) + j] = d_F[i * (dj_max + 2) + j] - delta_t * (p[(i+1) * (dj_max + 2) + j] - p[i * (dj_max + 2) + j]) / delta_x;
        if (j <= dj_max - 1) v[i * (dj_max + 1) + j] = d_G[i * (dj_max + 1) + j] - delta_t * (p[i * (dj_max + 2) + (j+1)] - p[i * (dj_max + 2) + j]) / delta_y;
    }
}

__global__ void extract_value_kernel(double* d_u, double* d_v, double* d_p, int i_max, j_max, double* result) {
    int idx = (i_max / 2) * (j_max + 2) + (j_max / 2);
    result[0] = d_u[idx];
    result[1] = d_v[idx];
    result[2] = d_p[idx];
}